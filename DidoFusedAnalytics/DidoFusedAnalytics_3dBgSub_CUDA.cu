#include "hip/hip_runtime.h"
/**  *****************************************************************************
* This program is the confidential and proprietary product of Overview    
* Limited. Any unauthorised use, reproduction or transfer of this         
* program is strictly prohibited.              
* Copyright 2017 Overview Limited. (Subject to limited                    
* distribution and restricted disclosure only.) All rights reserved.
*
* @file    	DidoFusedAnalytics_3dBgSub_CUDA.h
* @author  	SL
* @version 	1
* @date    	2017-06-29
* @brief   	class that does the background subtraction on the GPU
 *****************************************************************************
**/

/**
*	description of the algorithm
*	this uses a standard MOG2 based background subtractor with a few key differences -

*	it checks if the background point would be in range of the lidar, and if not, adjusts the thresholds appropriately and only uses thermal instead
*	if it is in range of the lidar,  it maintains separate variances for each component, and because the lidar values are returned as a range, it models them
*	as a gaussian with mean at the center of the range and standard deviation = half the width of the range. Then the values are based simply on the joint integrals of the two distributions 
*/



#include "global_defines.h"
#include "DidoFusedAnalytics_3dBgSub_CUDA.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include "math.h"
#include "hip/hip_math_constants.h"
#include "hip/hip_runtime_api.h"

//for our logging we will throw an error that can then be caught by the surrounding code that is allowed to include boost
#include "CUDA_Exception.h"


/*
   if the computer being used doesn't have a GPU, define DIDOLIDAR_NOGPU as 1 in the preprocessor, and this wil produce some noops instead. It still requires the nvidia sdk to compile at all, however
*/



#if DIDOLIDAR_NOGPU

#else
//error handling function
static void HandleError( hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
	//	hipDeviceReset();
		throw overview::CUDA_Exception(hipGetErrorString( err ) , err, line, file);
    }
}
#define HANDLE_ERROR(err) {HandleError((err), __FILE__, __LINE__);} 
#endif

namespace overview
{

namespace bgrcuda
{
#define NOPOINTVALUE -1.0f
	//convenience function for swapping with
	__device__ __forceinline__ void swap(float * array, int ind1, int ind2)
	{
		float tmp = array[ind1];
		array[ind1] = array[ind2];
		array[ind2] = tmp;
	}

	template<typename T>
	__global__ void downsample(const T * in, T* out, int nrows, int ncols, int scale)
	{
		const int totind_x = (threadIdx.x + blockIdx.x * blockDim.x);
		const int totind_y = (threadIdx.y + blockIdx.y * blockDim.y);
		if (totind_x >= ncols / scale || totind_y > nrows / scale) return;
		double total = 0;
		int npts = 0;
		for (int i = 0; i < scale; i++)
		{
			int index_x = totind_x*scale + i;
			if (index_x >= ncols) continue;
			for (int j = 0; j < scale; j++)
			{
				int index_y = totind_y*scale + j;
				if (index_y >= nrows) continue;
				total += in[index_x + ncols*index_y];
				npts++;
			}
		}
		out[totind_x + totind_y*(ncols / scale)] = (T)(total / npts);
	}

	__global__ void upsample(const float * in_pano, float * out_pano,int nrows,  int ncols, int scale)
	{
		int index_x = (threadIdx.x + blockIdx.x * blockDim.x);
		int index_y = (threadIdx.y + blockIdx.y * blockDim.y);
		if (index_x / scale < ncols / scale && index_y / scale < nrows / scale)
			out_pano[index_x + index_y*ncols] = in_pano[(index_x / scale) + (index_y / scale)*(ncols / scale)];
	}

	//the actual bgr
	__global__ void mixturegaussians(const float * ranges_min, const float * ranges_max, float * rangemdl, float * rangevar, const thermalType * temps,
		float * tempmdl, float * tempvar, float * modelweights, float* out_min, float* out_max, int rows, int cols, float alphaT, float alpha1, float prune, DidoFusedAnalytics_3dBgSub_CUDA::bgrPars pars)
	{
		const int x = blockIdx.x * blockDim.x + threadIdx.x;
		const int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x >= cols || y >= rows)
			return;

		float r_min = ranges_min[x + y*cols];
		float r_max = ranges_max[x + y*cols];
		float r_width = r_max - r_min;
		float r_center =  (r_max + r_min)/2;
		float temp = temps[x + y*cols];

		//check if the observation has a range
		bool hasRange = r_center > 0;


		//calculate distances to the modes (+ sort)
		//here we need to go in descending order!!!

		bool background = false; // true - the pixel classified as background

		//internal:

		bool fitsPDF = false; //if it remains zero a new GMM mode will be added

		float totalWeight = 0.0f;

		//go through all modes
		int lastmode = 0;
		for (int mode = 0; mode < pars.c_nmixtures; ++mode)
		{
			int modeind = (x + (y*cols))*pars.c_nmixtures + mode;
			//skip modes with no weight
			//need only weight if fit is found
			if (modelweights[modeind] <= 0) continue;
			float weight = alpha1 * modelweights[modeind] + prune;
			lastmode++;
			//fit not found yet
			if (!fitsPDF)
			{
				bool hasModelRange = rangemdl[modeind] > 0;
				//check if it belongs to some of the remaining modes
				float t_var = tempvar[modeind];
				//our observations of range are also gaussian distibuted, so we look at the distribution of the convolution
				float r_var = rangevar[modeind] + (r_width*r_width);
            
				//calculate difference and distance
				float t_diff = tempmdl[modeind] - temp;
				float r_diff = rangemdl[modeind] - r_center;
				//weighted distance in both directions
				float dist2 = hasRange && hasModelRange ? t_diff*t_diff*r_var + r_diff*r_diff*t_var : t_diff*t_diff;
				float bgthresh = hasRange && hasModelRange ? pars.c_Tb * t_var * r_var : pars.c_Tb*t_var;
				float genthresh = hasRange && hasModelRange ? pars.c_Tg * t_var * r_var : pars.c_Tg*t_var;

				//background? - Tb - usually larger than Tg
				if (totalWeight < pars.c_TB && dist2 < bgthresh)
					background = true;

				//check fit
				if (dist2 < genthresh)
				{
					//belongs to the mode
					fitsPDF = true;

					//update distribution

					//update weight
					weight += alphaT;
					float k = alphaT / weight;

					//update variance
					float t_varnew = t_var + k * (t_diff*t_diff - t_var);
					//integrating the weighting against the probability of the observation
					float r_varnew = rangevar[modeind] + hasRange && hasModelRange ? k * ((r_width*r_width + 1)*(r_diff*r_diff) + pars.c_r_varInflate - rangevar[modeind]) : 0;

					//update means
					tempmdl[modeind] = tempmdl[modeind] - k * t_diff;
					rangemdl[modeind] = hasModelRange ? (rangemdl[modeind] - hasRange ? k *( r_diff ) : 0) : r_center;


					//limit the variance
					t_varnew = (t_varnew < pars.c_varMin_t) ? pars.c_varMin_t : (t_varnew > pars.c_varMax_t)? pars.c_varMax_t : t_varnew;
					r_varnew = (r_varnew < pars.c_varMin_r) ? pars.c_varMin_r : (r_varnew > pars.c_varMax_r)? pars.c_varMax_r : r_varnew;

					rangevar[modeind] = r_varnew;
					tempvar[modeind] = t_varnew;

					//sort
					//all other weights are at the same place and
					//only the matched (iModes) is higher -> just find the new place for it

					for (int i = mode; i > 0; --i)
					{
						//check one up
						if (weight < modelweights[(i - 1) + pars.c_nmixtures*(x + y*cols)])
							break;

						//swap one up
						swap(modelweights, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
						swap(rangevar, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
						swap(tempvar, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
						swap(rangemdl, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
						swap(tempmdl, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
					}

					//belongs to the mode - bFitsPDF becomes 1
				}
			} // !fitsPDF

			//check prune
			if (weight < -prune)
			{
				weight = 0.0f;
				lastmode--;
			}

			modelweights[modeind] = weight; //update weight by the calculated value
			totalWeight += weight;
		}

		//renormalize weights

		totalWeight = totalWeight == 0 ? 1.f : 1.f / totalWeight;
		for (int mode = 0; mode < pars.c_nmixtures; ++mode)
			modelweights[(x + (y*cols))*pars.c_nmixtures + mode] *= totalWeight;

		//make new mode if needed and exit

		if (!fitsPDF)
		{
			if(lastmode == pars.c_nmixtures) lastmode--;
			if (lastmode == 0)
				modelweights[(x + (y*cols))*pars.c_nmixtures + lastmode] = 1.f;
			else
			{
				modelweights[(x + (y*cols))*pars.c_nmixtures + lastmode] = alphaT;

				// renormalize all other weights

				for (int i = lastmode - 1; i >= 0 ; i--)
					modelweights[(x + (y*cols))*pars.c_nmixtures + i] *= alpha1;
			}

			// init

			rangemdl[(x + (y*cols))*pars.c_nmixtures + lastmode] = hasRange ? r_center : -1.0f;
			tempmdl[(x + (y*cols))*pars.c_nmixtures + lastmode] = temp;
			tempvar[(x + (y*cols))*pars.c_nmixtures + lastmode] = pars.c_varInit_t;
			rangevar[(x + (y*cols))*pars.c_nmixtures + lastmode] = pars.c_varInit_r;

			//sort
			//find the new place for it

			for (int i = lastmode - 1; i > 0; --i)
			{
				// check one up
				if (alphaT < modelweights[(i - 1) + pars.c_nmixtures*(x + y*cols)])
					break;

				//swap one up
				swap(modelweights, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
				swap(rangevar, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
				swap(tempvar, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
				swap(rangemdl, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
				swap(tempmdl, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
			}
		}
		//return inf if we don't have a range
		out_min[x + y*cols] = background ? NOPOINTVALUE : hasRange ? r_min : HIP_INF_F;
		out_max[x + y*cols] = background ? NOPOINTVALUE : hasRange ? r_max : HIP_INF_F;
	}

	__global__ void mixturegaussians_onlyTherm(const float * ranges_min, const float * ranges_max, const thermalType * temps, float * tempmdl, float * tempvar, 
		float * modelweights, float* out_min, float* out_max, int rows, int cols, float alphaT, float alpha1, float prune, DidoFusedAnalytics_3dBgSub_CUDA::bgrPars pars)
	{
		const int x = blockIdx.x * blockDim.x + threadIdx.x;
		const int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x >= cols || y >= rows)
			return;

		float r_min = ranges_min[x + y*cols];
		float r_max = ranges_max[x + y*cols];
		float temp = temps[x + y*cols];

		//calculate distances to the modes (+ sort)
		//here we need to go in descending order!!!

		bool background = false; // true - the pixel classified as background

		//internal:

		bool fitsPDF = false; //if it remains zero a new GMM mode will be added

		float totalWeight = 0.0f;

		//go through all modes
		int lastmode = 0;
		for (int mode = 0; mode < pars.c_nmixtures; ++mode)
		{
			int modeind = (x + (y*cols))*pars.c_nmixtures + mode;
			//skip modes with no weight
			//need only weight if fit is found
			if (modelweights[modeind] <= 0) continue;
			float weight = alpha1 * modelweights[modeind] + prune;
			lastmode++;
			//fit not found yet
			if (!fitsPDF)
			{
				//check if it belongs to some of the remaining modes
				float t_var = tempvar[modeind];
            
				//calculate difference and distance
				float t_diff = tempmdl[modeind] - temp;
				//weighted distance in both directions
				float dist2 = t_diff*t_diff;
				float bgthresh = pars.c_Tb*t_var;
				float genthresh =  pars.c_Tg*t_var;

				//background? - Tb - usually larger than Tg
				if (totalWeight < pars.c_TB && dist2 < bgthresh)
					background = true;

				//check fit
				if (dist2 < genthresh)
				{
					//belongs to the mode
					fitsPDF = true;

					//update distribution

					//update weight
					weight += alphaT;
					float k = alphaT / weight;

					//update variance
					float t_varnew = t_var + k * (t_diff*t_diff - t_var);
					//integrating the weighting against the probability of the observation

					//update means
					tempmdl[modeind] = tempmdl[modeind] - k * t_diff;
					
					//limit the variance
					t_varnew = (t_varnew < pars.c_varMin_t) ? pars.c_varMin_t : (t_varnew > pars.c_varMax_t)? pars.c_varMax_t : t_varnew;

					tempvar[modeind] = t_varnew;

					//sort
					//all other weights are at the same place and
					//only the matched (iModes) is higher -> just find the new place for it

					for (int i = mode; i > 0; --i)
					{
						//check one up
						if (weight < modelweights[(i - 1) + pars.c_nmixtures*(x + y*cols)])
							break;

						//swap one up
						swap(modelweights, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
						swap(tempvar, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
						swap(tempmdl, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
					}

					//belongs to the mode - bFitsPDF becomes 1
				}
			} // !fitsPDF

			//check prune
			if (weight < -prune)
			{
				weight = 0.0f;
				lastmode--;
			}

			modelweights[modeind] = weight; //update weight by the calculated value
			totalWeight += weight;
		}

		//renormalize weights

		totalWeight = totalWeight == 0 ? 1.f : 1.f / totalWeight;
		for (int mode = 0; mode < pars.c_nmixtures; ++mode)
			modelweights[(x + (y*cols))*pars.c_nmixtures + mode] *= totalWeight;

		//make new mode if needed and exit

		if (!fitsPDF)
		{
			if(lastmode == pars.c_nmixtures) lastmode--;
			if (lastmode == 0)
				modelweights[(x + (y*cols))*pars.c_nmixtures + lastmode] = 1.f;
			else
			{
				modelweights[(x + (y*cols))*pars.c_nmixtures + lastmode] = alphaT;

				// renormalize all other weights

				for (int i = lastmode - 1; i >= 0 ; i--)
					modelweights[(x + (y*cols))*pars.c_nmixtures + i] *= alpha1;
			}

			// init

			tempmdl[(x + (y*cols))*pars.c_nmixtures + lastmode] = temp;
			tempvar[(x + (y*cols))*pars.c_nmixtures + lastmode] = pars.c_varInit_t;

			//sort
			//find the new place for it

			for (int i = lastmode - 1; i > 0; --i)
			{
				// check one up
				if (alphaT < modelweights[(i - 1) + pars.c_nmixtures*(x + y*cols)])
					break;

				//swap one up
				swap(modelweights, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
				swap(tempvar, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
				swap(tempmdl, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
			}
		}
		//return inf if we don't have a range
		out_min[x + y*cols] = background ? NOPOINTVALUE : r_min;
		out_max[x + y*cols] = background ? NOPOINTVALUE : r_max;
	}

#define REGION_WIDTH 3
#define REGION_AREA (REGION_WIDTH*REGION_WIDTH)
#define REGION_HWIDTH (REGION_WIDTH/2)

	//the actual bgr - region based method
	__global__ void mixturegaussians_region(const float * ranges_min, const float * ranges_max, float * rangemdl, float * rangevar, const thermalType * temps,
		float * tempmdl, float * tempvar, float * modelweights, float* out_min, float* out_max, int rows, int cols, float alphaT, float alpha1, float prune, DidoFusedAnalytics_3dBgSub_CUDA::bgrPars pars)
	{
		const int x = blockIdx.x * blockDim.x + threadIdx.x;
		const int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x >= cols || y >= rows)
			return;

		float r_center[REGION_AREA], r_width =0, temp[REGION_AREA];
		for (int i = 0; i < REGION_AREA; i++)
		{
			int sx = x - REGION_HWIDTH + i % REGION_WIDTH;
			int sy = y - REGION_HWIDTH + i / REGION_WIDTH;
			if (sx < 0 || sy < 0 || sx >= cols || sy >= rows)
			{
				r_center[i] = 0;
				temp[i] = 0;
			}
			else
			{
				float r_min = ranges_min[sx + sy*cols];
				float r_max = ranges_max[sx + sy*cols];
				r_width += r_max - r_min;
				r_center[i] = (r_max + r_min) / 2;
				temp[i] = temps[sx + sy*cols];
			}
		}
		//normalise the variance of the range observations
		r_width /= REGION_AREA;
		//check if the observation is outside our maximum range		
		bool hasRange = r_center[4] > 0;


		//calculate distances to the modes (+ sort)
		//here we need to go in descending order!!!

		bool background = false; // true - the pixel classified as background

		//internal:

		bool fitsPDF = false; //if it remains zero a new GMM mode will be added

		float totalWeight = 0.0f;

		//go through all modes
		int lastmode = 0;
		for (int mode = 0; mode < pars.c_nmixtures; ++mode)
		{
			int modeind = (x + (y*cols))*pars.c_nmixtures + mode;
			//skip modes with no weight
			//need only weight if fit is found
			if (modelweights[modeind] <= 0) continue;
			float weight = alpha1 * modelweights[modeind] + prune;
			lastmode++;
			//fit not found yet
			if (!fitsPDF)
			{
				bool hasModelRange = rangemdl[modeind* REGION_AREA + REGION_WIDTH] > 0;
				//check if it belongs to some of the remaining modes
				float t_var = tempvar[modeind];
				//our observations of range are also gaussian distibuted, so we look at the distribution of the convolution
				float r_var = rangevar[modeind] + (r_width*r_width);
            
				//calculate difference and distance
				float t_diff[REGION_AREA], r_diff[REGION_AREA];
				for (int i = 0; i < REGION_AREA; i++)
				{
					t_diff[i] = tempmdl[modeind*REGION_AREA + i] - temp[i];
					r_diff[i] = rangemdl[modeind * REGION_AREA + i] - r_center[i];
				}

				//clculate the L2norm of the adjusted differences
				float tdist = 0, rdist = 0;
				for (int i = 0; i < REGION_AREA; i++)
				{
					tdist += fmin(t_diff[i] * t_diff[i], pars.c_errrorCap * t_var);
					rdist += fmin(r_diff[i] * r_diff[i], pars.c_errrorCap * r_var);
				}
				float dist2 = hasRange && hasModelRange ? tdist*r_var + rdist*t_var : tdist;
				//weighted distance in both directions
				float bgthresh = hasRange && hasModelRange ? pars.c_Tb * t_var * r_var *REGION_AREA : pars.c_Tb*t_var *REGION_AREA;
				float genthresh = hasRange && hasModelRange ? pars.c_Tg * t_var * r_var * REGION_AREA : pars.c_Tg*t_var *REGION_AREA;

				//background? - Tb - usually larger than Tg
				if (totalWeight < pars.c_TB && dist2 < bgthresh)
					background = true;

				//check fit
				if (dist2 < genthresh)
				{
					//belongs to the mode
					fitsPDF = true;

					//update distribution

					//update weight
					weight += alphaT;
					float k = alphaT / weight;

					//update variance
					float t_varnew = t_var + k * (tdist/ REGION_AREA + pars.c_t_varInflate - t_var);
					//integrating the weighting against the probability of the observation
					float r_varnew = rangevar[modeind] + hasRange && hasModelRange ? (k * ((r_width*r_width + 1)*(rdist/ REGION_AREA) + pars.c_r_varInflate - rangevar[modeind])) : 0;

					//update means
					for (int i = 0; i < REGION_AREA; i++)
					{
						tempmdl[modeind*REGION_AREA +i] = tempmdl[modeind * REGION_AREA + i] - k * t_diff[i];
						rangemdl[modeind*REGION_AREA + i] = rangemdl[modeind * REGION_AREA + i] > 0 ? rangemdl[modeind * REGION_AREA + i] - k *(r_diff[i]) : r_center[i];
					}


					//limit the variance
					t_varnew = (t_varnew < pars.c_varMin_t) ? pars.c_varMin_t : (t_varnew > pars.c_varMax_t)? pars.c_varMax_t : t_varnew;
					r_varnew = (r_varnew < pars.c_varMin_r) ? pars.c_varMin_r : (r_varnew > pars.c_varMax_r)? pars.c_varMax_r : r_varnew;

					rangevar[modeind] = r_varnew;
					tempvar[modeind] = t_varnew;

					//sort
					//all other weights are at the same place and
					//only the matched (iModes) is higher -> just find the new place for it

					for (int i = mode; i > 0; --i)
					{
						//check one up
						if (weight < modelweights[(i - 1) + pars.c_nmixtures*(x + y*cols)])
							break;

						//swap one up
						swap(modelweights, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
						swap(rangevar, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
						swap(tempvar, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));

						for (int k = 0; k < REGION_AREA; k++)
						{
							swap(rangemdl, ((i - 1) + pars.c_nmixtures*(x + y*cols)) * REGION_AREA + k, (i + pars.c_nmixtures*(x + y*cols)) * REGION_AREA + k);
							swap(tempmdl, ((i - 1) + pars.c_nmixtures*(x + y*cols)) *REGION_AREA + k, (i + pars.c_nmixtures*(x + y*cols)) * REGION_AREA + k);
						}
					}

					//belongs to the mode - bFitsPDF becomes 1
				}
			} // !fitsPDF

			//check prune
			if (weight < -prune)
			{
				weight = 0.0f;
				lastmode--;
			}

			modelweights[modeind] = weight; //update weight by the calculated value
			totalWeight += weight;
		}

		//renormalize weights

		totalWeight = totalWeight == 0 ? 1.f : 1.f / totalWeight;
		for (int mode = 0; mode < pars.c_nmixtures; ++mode)
			modelweights[(x + (y*cols))*pars.c_nmixtures + mode] *= totalWeight;

		//make new mode if needed and exit

		if (!fitsPDF)
		{
			if(lastmode == pars.c_nmixtures) lastmode--;
			if (lastmode == 0)
				modelweights[(x + (y*cols))*pars.c_nmixtures + lastmode] = 1.f;
			else
			{
				modelweights[(x + (y*cols))*pars.c_nmixtures + lastmode] = alphaT;

				// renormalize all other weights

				for (int i = lastmode - 1; i >= 0 ; i--)
					modelweights[(x + (y*cols))*pars.c_nmixtures + i] *= alpha1;
			}

			// init


			for (int k = 0; k < REGION_AREA; k++)
			{
				rangemdl[((x + (y*cols))*pars.c_nmixtures + lastmode)*REGION_AREA + k] = r_center[k];
				tempmdl[((x + (y*cols))*pars.c_nmixtures + lastmode)*REGION_AREA +k] = temp[k];
			}

			tempvar[(x + (y*cols))*pars.c_nmixtures + lastmode] = pars.c_varInit_t;
			rangevar[(x + (y*cols))*pars.c_nmixtures + lastmode] = pars.c_varInit_r;

			//sort
			//find the new place for it

			for (int i = lastmode - 1; i > 0; --i)
			{
				// check one up
				if (alphaT < modelweights[(i - 1) + pars.c_nmixtures*(x + y*cols)])
					break;

				//swap one up
				swap(modelweights, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
				swap(rangevar, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
				swap(tempvar, (i - 1) + pars.c_nmixtures*(x + y*cols),i + pars.c_nmixtures*(x + y*cols));
				for (int k = 0; k < REGION_AREA; k++)
				{
					swap(rangemdl, ((i - 1) + pars.c_nmixtures*(x + y*cols)) * REGION_AREA + k, (i + pars.c_nmixtures*(x + y*cols)) * REGION_AREA + k);
					swap(tempmdl, ((i - 1) + pars.c_nmixtures*(x + y*cols)) * REGION_AREA + k, (i + pars.c_nmixtures*(x + y*cols)) * REGION_AREA + k);
				}
			}
		}

		out_min[x + y*cols] = background ? NOPOINTVALUE :  hasRange ? ranges_min[x + y*cols]: HIP_INF_F;
		out_max[x + y*cols] = background ? NOPOINTVALUE : hasRange ? ranges_max[x + y*cols] : HIP_INF_F;

	}

}

//called whenever we change the constants 
static inline float getUnivarateThresh(float thresh)
{
	//these numbers are numerically estimated from the normal distributiobns in the range 1-6
	return pow(thresh, 4)*0.0016298f - pow(thresh, 3)*0.0080105f - pow(thresh, 2)*0.1293664f + thresh*1.3835517f - 0.6398407f;
}

DidoFusedAnalytics_3dBgSub_CUDA::DidoFusedAnalytics_3dBgSub_CUDA(DidoFusedAnalytics_3dBgSub_CUDA & cp): useRegion(cp.useRegion), scale(cp.scale)
{
	cols = cp.cols;
	rows = cp.rows;
	ct = cp.ct;
	history = cp.history;
	nsteps = cp.nsteps;
	pars = cp.pars;

	//cuda allocated variables
#if DIDOLIDAR_NOGPU

#else
	int modelsize = rows*cols*pars.c_nmixtures * sizeof(float);
	int regionsize = useRegion ? modelsize*REGION_AREA : modelsize;

	//allocate the model
	HANDLE_ERROR(hipMalloc(&rangeModel, regionsize));
	HANDLE_ERROR(hipMalloc(&rangevars, modelsize));
	HANDLE_ERROR(hipMalloc(&tempmodel, regionsize));
	HANDLE_ERROR(hipMalloc(&tempvars, modelsize));
	HANDLE_ERROR(hipMalloc(&modelweights, modelsize));

	HANDLE_ERROR(hipMemcpy(modelweights,cp.modelweights, modelsize,hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(rangeModel, cp.rangeModel, regionsize, hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(rangevars, cp.rangevars, modelsize, hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(tempmodel, cp.tempmodel, regionsize, hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(tempvars, cp.tempvars, modelsize, hipMemcpyDeviceToDevice));
#endif
}

DidoFusedAnalytics_3dBgSub_CUDA::DidoFusedAnalytics_3dBgSub_CUDA(DidoFusedAnalytics_3dBgSub_CUDA && mv): useRegion(mv.useRegion), scale(mv.scale)
{
	cols = mv.cols;
	rows = mv.rows;
	ct = mv.ct;
	history = mv.history;
	nsteps = mv.nsteps;
	pars = mv.pars;

	modelweights = mv.modelweights;
	mv.modelweights = nullptr;
	rangeModel = mv.rangeModel;
	mv.rangeModel = nullptr;
	rangevars = mv.rangevars;
	mv.rangevars = nullptr;
	tempmodel = mv.tempmodel;
	mv.tempmodel = nullptr;
	tempvars = mv.tempvars;
	mv.tempvars = nullptr;

}

DidoFusedAnalytics_3dBgSub_CUDA::DidoFusedAnalytics_3dBgSub_CUDA(int _rows, int _cols, bool useregion, int _scale)
	: rows(_rows), cols(_cols), useRegion(useregion), scale(_scale)
{
   #if DIDOLIDAR_NOGPU

   #else
	int modelsize = (rows/scale)*(cols/scale)*pars.c_nmixtures*sizeof(float);
	int regionsize = useRegion ? modelsize*REGION_AREA : modelsize;
	//allocate the model
	HANDLE_ERROR(hipMalloc(&rangeModel, regionsize));
	HANDLE_ERROR(hipMalloc(&rangevars, modelsize));
	HANDLE_ERROR(hipMalloc(&tempmodel, regionsize));
	HANDLE_ERROR(hipMalloc(&tempvars, modelsize));
	HANDLE_ERROR(hipMalloc(&modelweights, modelsize));

	pars.c_Tb_u = getUnivarateThresh(pars.c_Tb);
	pars.c_Tg_u = getUnivarateThresh(pars.c_Tg);
	#endif
}

DidoFusedAnalytics_3dBgSub_CUDA::~DidoFusedAnalytics_3dBgSub_CUDA()
{

   #if DIDOLIDAR_NOGPU

   #else
	//deallocate the model
	if(rangeModel != nullptr) (hipFree(rangeModel));
	if(rangevars != nullptr) (hipFree(rangevars));
	if(tempmodel != nullptr) (hipFree(tempmodel));
	if(tempvars != nullptr) (hipFree(tempvars));
	if(modelweights != nullptr) (hipFree(modelweights));
	#endif
}

void DidoFusedAnalytics_3dBgSub_CUDA::apply(const thermalType * input_t, 
	const float * input_d_min, const float * input_d_max, float * out_min, float * out_max, float learningRate) 
{

#if DIDOLIDAR_NOGPU

#else
	nsteps++;
	float lr;
	//allocate the learning rate
	if(learningRate < 0)
	{
		lr = nsteps > history ? 1.f/history : 1.f/(nsteps);
	}
	else
	{
		lr = learningRate;
	}

	//downsample it
	float * l_d_min, *l_d_max, *l_o_max, *l_o_min;
	thermalType * l_therm;
	HANDLE_ERROR(hipMalloc(&l_d_max, (cols / scale)*(rows / scale) * sizeof(float)));
	HANDLE_ERROR(hipMalloc(&l_d_min, (cols / scale)*(rows / scale) * sizeof(float)));
	HANDLE_ERROR(hipMalloc(&l_o_max, (cols / scale)*(rows / scale) * sizeof(float)));
	HANDLE_ERROR(hipMalloc(&l_o_min, (cols / scale)*(rows / scale) * sizeof(float)));
	HANDLE_ERROR(hipMalloc(&l_therm, (cols / scale)*(rows / scale) * sizeof(thermalType)));

	int blockdim = 16;
	dim3 grid((cols / scale) / blockdim + 1, (rows / scale) / blockdim + 1);
	dim3 block(blockdim, blockdim);
	if (scale == 1)
	{
		HANDLE_ERROR(hipMemcpy(l_d_max, input_d_max, cols*rows*sizeof(float), hipMemcpyDeviceToDevice));
		HANDLE_ERROR(hipMemcpy(l_d_min, input_d_min, cols*rows * sizeof(float), hipMemcpyDeviceToDevice));
	}
	else
	{
		bgrcuda::downsample<float> <<<grid, block >>> (input_d_max, l_d_max, rows, cols, scale);
		bgrcuda::downsample <float><<<grid, block >>> (input_d_min, l_d_min, rows, cols, scale);
		bgrcuda::downsample <thermalType> <<<grid, block >>> (input_t, l_therm, rows, cols, scale);
	}
	hipDeviceSynchronize();
	HANDLE_ERROR(hipGetLastError());

	//run the bgr
	if(useRegion)
	{
		bgrcuda::mixturegaussians_region<<<grid, block>>>(l_d_min, l_d_max, rangeModel, rangevars, l_therm,
			tempmodel, tempvars, modelweights, l_o_min, l_o_max, rows/scale, cols/scale, lr, 1.0f - lr, -lr*ct, pars);
	}
	else
	{
		bgrcuda::mixturegaussians<<<grid, block>>>(l_d_min, l_d_max, rangeModel, rangevars, l_therm,
			tempmodel, tempvars, modelweights, l_o_min, l_o_max, rows/scale, cols/scale, lr, 1.0f - lr, -lr*ct, pars);
	}
	hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());

	//upsample it
	if (scale == 1)
	{
		HANDLE_ERROR(hipMemcpy(out_max, l_o_max, cols*rows * sizeof(float), hipMemcpyDeviceToDevice));
		HANDLE_ERROR(hipMemcpy(out_min, l_o_min, cols*rows * sizeof(float), hipMemcpyDeviceToDevice));
	}
	else
	{
		grid = dim3((cols) / blockdim + 1, (rows ) / blockdim + 1);
		bgrcuda::upsample <<<grid, block >>> (l_o_max, out_max, rows, cols, scale);
		bgrcuda::upsample<<<grid, block >>> (l_o_min, out_min, rows, cols, scale);
	}
	hipDeviceSynchronize();
	HANDLE_ERROR(hipGetLastError());

	if (l_o_max) hipFree(l_o_max);
	if (l_d_max) hipFree(l_d_max);
	if (l_d_min) hipFree(l_d_min);
	if (l_o_min) hipFree(l_o_min);
	if (l_therm) hipFree(l_therm);

#endif

}

void DidoFusedAnalytics_3dBgSub_CUDA::setHistory(int hist_)
{
	history = hist_;
}
void DidoFusedAnalytics_3dBgSub_CUDA::setBackgroundWeight(float TB)
{
	pars.c_TB = TB;
}

void DidoFusedAnalytics_3dBgSub_CUDA::setVariance(float initTempVar, float initRangeVar)
{
	//parameters for thermal space
	pars.c_varInit_t = initTempVar; // initial variance for new components
	pars.c_varMax_t = 5.0f * pars.c_varInit_t;
	pars.c_varMin_t = pars.c_varInit_t/1.5f;

	//params for range space
	pars.c_varInit_r = initRangeVar; // initial variance for new components
	pars.c_varMax_r = 5.0f * pars.c_varInit_r;
	pars.c_varMin_r = pars.c_varInit_r/4;

	pars.c_r_varInflate = initRangeVar / 3;
	pars.c_t_varInflate = initTempVar/3;
}
void DidoFusedAnalytics_3dBgSub_CUDA::setThresholds(float backgroundThresh, float generativeThresh)
{
	pars.c_Tb = backgroundThresh;
	pars.c_Tg  = generativeThresh;

	pars.c_Tb_u = getUnivarateThresh(pars.c_Tb);
	pars.c_Tg_u = getUnivarateThresh(pars.c_Tg);
}

DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly::DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly(DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly & cp): scale(cp.scale)
{
	cols = cp.cols;
	rows = cp.rows;
	ct = cp.ct;
	history = cp.history;
	nsteps = cp.nsteps;
	pars = cp.pars;

	//cuda allocated variables
#if DIDOLIDAR_NOGPU

#else
	int modelsize = rows*cols*pars.c_nmixtures * sizeof(float);
	int regionsize = modelsize;

	//allocate the model
	HANDLE_ERROR(hipMalloc(&tempmodel, regionsize));
	HANDLE_ERROR(hipMalloc(&tempvars, modelsize));
	HANDLE_ERROR(hipMalloc(&modelweights, modelsize));
	HANDLE_ERROR(hipMemcpy(modelweights,cp.modelweights, modelsize,hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(tempmodel, cp.tempmodel, regionsize, hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(tempvars, cp.tempvars, modelsize, hipMemcpyDeviceToDevice));
#endif
}

DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly::DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly(DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly && mv): scale(mv.scale)
{
	cols = mv.cols;
	rows = mv.rows;
	ct = mv.ct;
	history = mv.history;
	nsteps = mv.nsteps;
	pars = mv.pars;

	modelweights = mv.modelweights;
	mv.modelweights = nullptr;
	tempmodel = mv.tempmodel;
	mv.tempmodel = nullptr;
	tempvars = mv.tempvars;
	mv.tempvars = nullptr;

}

DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly::DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly(int _rows, int _cols, int _scale)
	: rows(_rows), cols(_cols),  scale(_scale)
{
   #if DIDOLIDAR_NOGPU

   #else
	int modelsize = (rows/scale)*(cols/scale)*pars.c_nmixtures*sizeof(float);
	int regionsize = modelsize;
	//allocate the model
	HANDLE_ERROR(hipMalloc(&tempmodel, regionsize));
	HANDLE_ERROR(hipMalloc(&tempvars, modelsize));
	HANDLE_ERROR(hipMalloc(&modelweights, modelsize));

	pars.c_Tb_u = getUnivarateThresh(pars.c_Tb);
	pars.c_Tg_u = getUnivarateThresh(pars.c_Tg);
	#endif
}

DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly::~DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly()
{

   #if DIDOLIDAR_NOGPU

   #else
	//deallocate the model
	if(tempmodel != nullptr) (hipFree(tempmodel));
	if(tempvars != nullptr) (hipFree(tempvars));
	if(modelweights != nullptr) (hipFree(modelweights));
	#endif
}

void DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly::apply(const thermalType * input_t, 
	const float * input_d_min, const float * input_d_max, float * out_min, float * out_max, float learningRate) 
{

#if DIDOLIDAR_NOGPU

#else
	nsteps++;
	float lr;
	//allocate the learning rate
	if(learningRate < 0)
	{
		lr = nsteps > history ? 1.f/history : 1.f/(nsteps);
	}
	else
	{
		lr = learningRate;
	}

	//downsample it
	float * l_d_min, *l_d_max, *l_o_max, *l_o_min;
	thermalType * l_therm;
	HANDLE_ERROR(hipMalloc(&l_d_max, (cols / scale)*(rows / scale) * sizeof(float)));
	HANDLE_ERROR(hipMalloc(&l_d_min, (cols / scale)*(rows / scale) * sizeof(float)));
	HANDLE_ERROR(hipMalloc(&l_o_max, (cols / scale)*(rows / scale) * sizeof(float)));
	HANDLE_ERROR(hipMalloc(&l_o_min, (cols / scale)*(rows / scale) * sizeof(float)));
	HANDLE_ERROR(hipMalloc(&l_therm, (cols / scale)*(rows / scale) * sizeof(thermalType)));

	int blockdim = 16;
	dim3 grid((cols / scale) / blockdim + 1, (rows / scale) / blockdim + 1);
	dim3 block(blockdim, blockdim);
	if (scale == 1)
	{
		HANDLE_ERROR(hipMemcpy(l_d_max, input_d_max, cols*rows*sizeof(float), hipMemcpyDeviceToDevice));
		HANDLE_ERROR(hipMemcpy(l_d_min, input_d_min, cols*rows * sizeof(float), hipMemcpyDeviceToDevice));
	}
	else
	{
		bgrcuda::downsample<float> <<<grid, block >>> (input_d_max, l_d_max, rows, cols, scale);
		bgrcuda::downsample <float><<<grid, block >>> (input_d_min, l_d_min, rows, cols, scale);
		bgrcuda::downsample <thermalType> <<<grid, block >>> (input_t, l_therm, rows, cols, scale);
	}
	hipDeviceSynchronize();
	HANDLE_ERROR(hipGetLastError());


		bgrcuda::mixturegaussians_onlyTherm<<<grid, block>>>(l_d_min, l_d_max, l_therm,
			tempmodel, tempvars, modelweights, l_o_min, l_o_max, rows/scale, cols/scale, lr, 1.0f - lr, -lr*ct, pars);
	hipDeviceSynchronize();
    HANDLE_ERROR(hipGetLastError());

	//upsample it
	if (scale == 1)
	{
		HANDLE_ERROR(hipMemcpy(out_max, l_o_max, cols*rows * sizeof(float), hipMemcpyDeviceToDevice));
		HANDLE_ERROR(hipMemcpy(out_min, l_o_min, cols*rows * sizeof(float), hipMemcpyDeviceToDevice));
	}
	else
	{
		grid = dim3((cols) / blockdim + 1, (rows ) / blockdim + 1);
		bgrcuda::upsample <<<grid, block >>> (l_o_max, out_max, rows, cols, scale);
		bgrcuda::upsample<<<grid, block >>> (l_o_min, out_min, rows, cols, scale);
	}
	hipDeviceSynchronize();
	HANDLE_ERROR(hipGetLastError());

	if (l_o_max) hipFree(l_o_max);
	if (l_d_max) hipFree(l_d_max);
	if (l_d_min) hipFree(l_d_min);
	if (l_o_min) hipFree(l_o_min);
	if (l_therm) hipFree(l_therm);

#endif

}

void DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly::setHistory(int hist_)
{
	history = hist_;
}
void DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly::setBackgroundWeight(float TB)
{
	pars.c_TB = TB;
}

void DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly::setVariance(float initTempVar, float initRangeVar)
{
	//parameters for thermal space
	pars.c_varInit_t = initTempVar; // initial variance for new components
	pars.c_varMax_t = 5.0f * pars.c_varInit_t;
	pars.c_varMin_t = pars.c_varInit_t/1.5f;

	//params for range space
	pars.c_varInit_r = initRangeVar; // initial variance for new components
	pars.c_varMax_r = 5.0f * pars.c_varInit_r;
	pars.c_varMin_r = pars.c_varInit_r/4;

	pars.c_r_varInflate = initRangeVar / 3;
	pars.c_t_varInflate = initTempVar/3;
}
void DidoFusedAnalytics_3dBgSub_CUDA_ThermalOnly::setThresholds(float backgroundThresh, float generativeThresh)
{
	pars.c_Tb = backgroundThresh;
	pars.c_Tg  = generativeThresh;

	pars.c_Tb_u = getUnivarateThresh(pars.c_Tb);
	pars.c_Tg_u = getUnivarateThresh(pars.c_Tg);
}


}