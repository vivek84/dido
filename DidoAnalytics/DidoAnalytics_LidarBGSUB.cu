#include "hip/hip_runtime.h"
/**  *****************************************************************************
* This program is the confidential and proprietary product of Overview
* Limited. Any unauthorised use, reproduction or transfer of this
* program is strictly prohibited.
* Copyright 2017 Overview Limited. (Subject to limited
* distribution and restricted disclosure only.) All rights reserved.
*
* @file    	DidoAnalytics_LidarBGSUB.cu
* @author  	SL
* @version 	1
* @date    	2017-10-02
* @brief   	GPU based Background subtraction for the Lidar
*****************************************************************************
**/

/*
* Algorithm Description - This keeps a background model of the lidar points by first putting the data into a 2.5d represention clustering points in neary
* angular space into bins. Then each for each bin of the model that bin searches the neighbourhood of bins in the observation. If a point in the observation
* is within the fixed threshold, the model point is considered observed, else it is considered unobserved. Observed points have their weights increased,
* unobserved points have their weight decreased. Then for each point in the observation it searches the bins of the model in the neighbourhood. The highest weight
* of model points within the bin is mainatained. If this weight is above a threshold, the observation is considered to be a background point, else it is a foreground point.
* Then for each point that was observed that had no points in the model close to it, those points are added to the model. Finally the points in the model are sorted in
* weight order, and any model points with a weight below a threshold are discarded.
* The learning rate used for updating the weights follows the standard history pattern of 1./min(nframes, history).
*
* The clustering is done using DBSCAN. The points are again clustered into bins (this time broader), and each point searches it's neighbourhood of bins to decide if it is a core
* point. Then in the next step each point looks through it's neighbourhood and takes the lowest core parent index in that index, which is iteratively repeated several times.
* finally all points are allocated the roots of the resulting tree structure as a parent index, and then the clusters are formed into vectors on the CPU by a single insertion sort pass
*/

#include "global_defines.h"
#include "DidoAnalytics_LidarBGSUB.h"
#include "CUDA_Exception.h"
#include "hip/hip_math_constants.h"

#ifdef _WIN32
#include <ppl.h>
#include <concurrent_unordered_map.h>
#else
#include <unordered_map>
#endif


#define DEBUG_TIMINGS 0

#if DEBUG_TIMINGS
#include <chrono>
#include <iostream>
#endif


//error handling function
static void HandleError(hipError_t err, const char *file, int line) {
	if (err != hipSuccess) {
		//		hipDeviceReset();
		throw overview::CUDA_Exception(hipGetErrorString(err), err, line, file);
	}
}
#define HANDLE_ERROR(err) {HandleError((err), __FILE__, __LINE__);} 

//data freeing simplification
inline void safe_Free(void * cudadata)
{
	if (cudadata != nullptr)HANDLE_ERROR(hipFree(cudadata));
}

namespace overview
{
	namespace lbgsCUDA
	{
		//convenience function for swapping with
		__device__ void swap(float * array, int ind1, int ind2)
		{
			float tmp = array[ind1];
			array[ind1] = array[ind2];
			array[ind2] = tmp;
		}

#define TILT_DISTANCE 0.1f
		//as the lidar is in beams, it uses the actual distance in x y space, anda fixed multiplier on the distance in titlt space
		__device__ __forceinline__ float getDist(float p1, float t1, float r1, float p2, float t2, float r2)
		{
			__align__(8) float st1 = sinf(t1), st2 = sinf(t2);
			__align__(8) float tiltdist = TILT_DISTANCE * abs(t1 - t2);
			return r2 * r2*st2*st2 + r1 * r1*st1*st1 - 2 * r1*r2*st1*st2*cosf(p1 - p2) + tiltdist * tiltdist;
		}

		__global__ void collatePoints(LidarBin obsBox, DidoLidar_rangeData* obs, int npts, int nrows, int ncols, float binWidth, float binHeight)
		{
			//iterate over the input points
			const int index = threadIdx.x + blockIdx.x * blockDim.x;
			if (index < npts)
			{
				//work out which bin you should be in
				int x = (int)(obs[index].pan / binWidth) % ncols;
				int y = min(max((int)((obs[index].tilt - HIP_PIO4_F) / binHeight), 0), nrows - 1);
				unsigned int binind = atomicAdd(&obsBox.npts[x + y * ncols], 1);
				if (binind < LIDARBGSUB_MAX_BIN_PTS)
				{
					unsigned int oind = binind + LIDARBGSUB_MAX_BIN_PTS * (x + y * ncols);
					obsBox.points_pan[oind] = obs[index].pan;
					obsBox.points_tilt[oind] = obs[index].tilt;
					obsBox.points_range[oind] = obs[index].range;
				}
			}
		}
		//we only search one above and below in tilt

		//updates the current model weights
		__global__ void bgsubKernel_pt1(LidarBin bgmodel, LidarBin obs, float * variances,
			float * weights, float threshold, float mindist_init, int nrows, int ncols, float lr, int searchWidth)
		{
			//one block per bin, using parallel threads for improved operation
			if (blockIdx.x >= ncols || blockIdx.y >= nrows)
				return;
			//correct the npoints for our inputs
			const int index = blockIdx.x + blockIdx.y*ncols;
			const int ind_y = blockIdx.y;

			//first proceed through the model and incriment or decrement depending if they are observed or not
			if (threadIdx.x < bgmodel.npts[index])
			{
				bool unobserved = true;
				bool unoccluded = true;
				bool lineobs = false; //checking if the packet is in the dataset (and no lost to occlusions/general IP stuff)
				float mdlpt_pan = bgmodel.points_pan[index*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];
				float mdlpt_tilt = bgmodel.points_tilt[index*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];
				float mdlpt_range = bgmodel.points_range[index*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];
				//tilt values are fixed so there's no value to vertical searching
				float mindist = variances[(index)*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];

				for (int i_x = -searchWidth; i_x <= searchWidth; i_x++)
				{
					//wrapping
					int ind_x = (i_x + blockIdx.x + ncols) % ncols;

					for (int j = 0; j < obs.npts[ind_x + ind_y * ncols] && j < LIDARBGSUB_MAX_BIN_PTS; j++)
					{
						float dist = getDist(obs.points_pan[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j],
							obs.points_tilt[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j],
							obs.points_range[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j], mdlpt_pan, mdlpt_tilt, mdlpt_range);
						if (dist < mindist * 3)
						{
							//		mindist = mindist + lr*(dist - mindist);
							unobserved = false;
						}
						else
							//is it in the line at all?
						{
							if (abs(obs.points_pan[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j] - mdlpt_pan) < 0.0002f)
							{
								lineobs = true;
								//is somethin closer and at the angle?
								if ((obs.points_range[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j] + 0.5f < mdlpt_range) &&
									(abs(obs.points_tilt[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j] - mdlpt_tilt) < 0.0001f))
								{
									unoccluded = false;
								}
							}
						}
					}

				}
				//update the point appropriately
				if (!unobserved || (unoccluded && lineobs))
				{
					weights[(index)*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x] = weights[(index)*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x] * (1.0f - lr) + (unobserved ? 0 : lr);
					//update my variance

					variances[(index)*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x] = min(max(mindist, mindist_init / 3), mindist_init * 3);

				}
			}
		}

		__global__ void bgsubKernel_pt2(LidarBin bgmodel, LidarBin obs, DidoLidar_rangeData* output, int * noutput, float * variances,
			float * weights, bool * addToModel, float threshold, int nrows, int ncols, int searchWidth)
		{
			//one block per bin, using parallel threads for improved operation
			if (blockIdx.x >= ncols || blockIdx.y >= nrows)
				return;
			//correct the npoints for our inputs
			const int index = blockIdx.x + blockIdx.y*ncols;
			const int ind_y = blockIdx.y;

			//then go through the observations and see if they are background and whether they are new
			//this does duplicate effort, but is needed to keep parallel determinism

			if (threadIdx.x < obs.npts[index])
			{
				float obsweight = -1.0f;
				bool newpoint = true;
				float obspt_pan = obs.points_pan[index*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];
				float obspt_tilt = obs.points_tilt[index*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];
				float obspt_range = obs.points_range[index*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];

				for (int i_x = -searchWidth; i_x <= searchWidth; i_x++)
				{
					//wrapping
					int ind_x = (i_x + blockIdx.x + ncols) % ncols;

					for (int j = 0; j < bgmodel.npts[ind_x + ind_y * ncols] && j < LIDARBGSUB_MAX_BIN_PTS; j++)
					{
						float dist = getDist(bgmodel.points_pan[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j],
							bgmodel.points_tilt[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j],
							bgmodel.points_range[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j],
							obspt_pan, obspt_tilt, obspt_range);
						float mindist = variances[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j];
						if (dist < 4 * mindist)
						{
							obsweight = max(obsweight, weights[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j]);
							if (dist < mindist * 3)
							{
								newpoint = false;
								//break;///its a sorted list (but this makes it slower due to awkwardness)
							}
						}
					}
				}
				//update the point appropriately

				//mark the point for output
				addToModel[index*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x] = newpoint;

				if (obsweight < threshold)
				{
					//put it in the output
					int outind = atomicAdd(noutput, 1);
					output[outind].range = obspt_range;
					output[outind].pan = obspt_pan;
					output[outind].tilt = obspt_tilt;
				}
			}

		}

		//sorts the models and culls the unobserved points and ones too close to each other
		__global__ void sortModels(LidarBin bgmodel, LidarBin obs, bool *addToModel, float * variances, float * weights, int nrows, int ncols, float lr, float mindist_init)
		{
			if (blockIdx.x >= ncols || blockIdx.y >= nrows) return;

			const int idx = (blockIdx.x + blockIdx.y*ncols);

			//produce an insertion vector
			__shared__ bool stillvalid[LIDARBGSUB_MAX_BIN_PTS];
			if (threadIdx.x < obs.npts[idx] && addToModel[idx*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x])
			{
				stillvalid[threadIdx.x] = true;
			}
			else stillvalid[threadIdx.x] = false;
			__syncthreads();
			//reduce it own by removing close vectors
			for (int i = 1; i < LIDARBGSUB_MAX_BIN_PTS / 2; i++)	//step size
			{
				int fidx = threadIdx.x + i * (threadIdx.x / i);
				int sidx = fidx + i;
				if (fidx < obs.npts[idx] && sidx < obs.npts[idx])
				{
					//compare and coalesc
					if (stillvalid[fidx] && stillvalid[sidx] && getDist(obs.points_pan[idx*LIDARBGSUB_MAX_BIN_PTS + fidx],
						obs.points_tilt[idx*LIDARBGSUB_MAX_BIN_PTS + fidx], obs.points_range[idx*LIDARBGSUB_MAX_BIN_PTS + fidx],
						obs.points_pan[idx*LIDARBGSUB_MAX_BIN_PTS + sidx], obs.points_tilt[idx*LIDARBGSUB_MAX_BIN_PTS + sidx], obs.points_range[idx*LIDARBGSUB_MAX_BIN_PTS + sidx]) < mindist_init)
					{
						stillvalid[sidx] = false;
					}
				}
				__syncthreads();
			}


			//then insert the remaining ones

			if (stillvalid[threadIdx.x])
			{
				//inputs  he new points here
				unsigned int npts = atomicAdd(&bgmodel.npts[idx], 1);
				if (npts < LIDARBGSUB_MAX_BIN_PTS)
				{
					int lidx = (idx)* LIDARBGSUB_MAX_BIN_PTS + npts;
					//add it to the model
					bgmodel.points_pan[lidx] = obs.points_pan[idx*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];
					bgmodel.points_tilt[lidx] = obs.points_tilt[idx*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];
					bgmodel.points_range[lidx] = obs.points_range[idx*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];
					weights[lidx] = lr;
					variances[lidx] = mindist_init;
				}
			}
			__syncthreads();
			//reset to max if we overflow
			if (threadIdx.x == 0) bgmodel.npts[idx] = min(bgmodel.npts[idx], (unsigned int)LIDARBGSUB_MAX_BIN_PTS);

			__syncthreads();

			// bubble sort the models (in parallel) so we can remove the worse
			__shared__ bool swapped;
			if (threadIdx.x == 0) swapped = true;
			while (swapped)
			{
				if (threadIdx.x == 0) swapped = false;
				if (threadIdx.x < bgmodel.npts[idx] / 2)
				{
					int sidx1 = idx * LIDARBGSUB_MAX_BIN_PTS + threadIdx.x * 2 + 1;
					int sidx2 = idx * LIDARBGSUB_MAX_BIN_PTS + threadIdx.x * 2;
					if (weights[sidx1] > weights[sidx2])
					{
						swap(weights, sidx1, sidx2);
						swap(variances, sidx1, sidx2);
						swap(bgmodel.points_pan, sidx1, sidx2);
						swap(bgmodel.points_tilt, sidx1, sidx2);
						swap(bgmodel.points_range, sidx1, sidx2);
						swapped = true;
					}

					sidx1 = idx * LIDARBGSUB_MAX_BIN_PTS + threadIdx.x * 2 + 2;
					sidx2 = idx * LIDARBGSUB_MAX_BIN_PTS + threadIdx.x * 2 + 1;
					if (threadIdx.x * 2 + 2 < bgmodel.npts[idx] && weights[sidx1] > weights[sidx2])
					{
						swap(weights, sidx1, sidx2);
						swap(variances, sidx1, sidx2);
						swap(bgmodel.points_pan, sidx1, sidx2);
						swap(bgmodel.points_tilt, sidx1, sidx2);
						swap(bgmodel.points_range, sidx1, sidx2);
						swapped = true;
					}
				}
				__syncthreads();
			}
			__syncthreads();
			//now remove any that have negative weights
			if (threadIdx.x < bgmodel.npts[idx])
			{
				if (weights[idx*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x] < lr / 4) atomicDec(&bgmodel.npts[idx], 0);
			}
		}

#define NO_POINT_PARENT -2
#define NON_CORE_PARENT -1

		__global__ void PDSCAN_init(const LidarBin fgpts, int* parents, bool *core, int nrows, int ncols, float mindist, int ncore, int searchwidth)
		{
			if (blockIdx.x >= ncols || blockIdx.y >= nrows) return;
			const int idx = (blockIdx.x + blockIdx.y*ncols);
			const int pind = idx * LIDARBGSUB_MAX_BIN_PTS + threadIdx.x;
			if (threadIdx.x < fgpts.npts[idx])
			{
				//search your neighbourhood to see how many neighbours you gave
				int nneighbours = 0;

				float obspt_pan = fgpts.points_pan[idx*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];
				float obspt_tilt = fgpts.points_tilt[idx*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];
				float obspt_range = fgpts.points_range[idx*LIDARBGSUB_MAX_BIN_PTS + threadIdx.x];

				for (int ind_y = (int)(blockIdx.y) - 2; ind_y <= blockIdx.y + 2; ind_y++)
				{
					if (ind_y >= 0 && ind_y < nrows)
					{
						for (int i_x = -searchwidth; i_x <= searchwidth; i_x++)
						{
							//wrapping
							int ind_x = (i_x + blockIdx.x + ncols) % ncols;

							for (int j = 0; j < fgpts.npts[ind_x + ind_y * ncols] && j < LIDARBGSUB_MAX_BIN_PTS; j++)
							{
								if (getDist(obspt_pan, obspt_tilt, obspt_range,
									fgpts.points_pan[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j],
									fgpts.points_tilt[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j],
									fgpts.points_range[(ind_x + ind_y * ncols)*LIDARBGSUB_MAX_BIN_PTS + j]) < mindist)
								{
									nneighbours++;
								}
							}
						}

					}
				}
				if (nneighbours >= ncore)
				{

					core[pind] = true;
					parents[pind] = pind;
				}
				else
				{
					core[pind] = false;
					parents[pind] = NON_CORE_PARENT;
				}
			}
			else if (threadIdx.x < LIDARBGSUB_MAX_BIN_PTS)
			{
				core[pind] = false;
				parents[pind] = NO_POINT_PARENT;
			}
		}

		__global__ void PDSCAN_local(const LidarBin fgpts, const int * parentsin, int * parentsout, const bool *core, int nrows, int ncols, float epsilon, int searchwidth)
		{
			if (blockIdx.x >= ncols || blockIdx.y >= nrows) return;
			const int idx = (blockIdx.x + blockIdx.y*ncols);
			const int pind = idx * LIDARBGSUB_MAX_BIN_PTS + threadIdx.x;
			if (threadIdx.x < fgpts.npts[idx])
			{
				//check every point in your region to populate your neigbourhood vector

				int my_parent = parentsin[pind];
				float obspt_pan = fgpts.points_pan[pind];
				float obspt_tilt = fgpts.points_tilt[pind];
				float obspt_range = fgpts.points_range[pind];

				for (int ind_y = (int)(blockIdx.y) - 2; ind_y <= blockIdx.y + 2; ind_y++)
				{
					if (ind_y >= 0 && ind_y < nrows)
					{
						for (int i_x = -searchwidth; i_x <= searchwidth; i_x++)
						{
							//wrapping
							int s_indx = ind_y * ncols + ((i_x + blockIdx.x + ncols) % ncols);

							for (int j = 0; j < fgpts.npts[s_indx] && j < LIDARBGSUB_MAX_BIN_PTS; j++)
							{
								if (core[s_indx*LIDARBGSUB_MAX_BIN_PTS + j] && getDist(obspt_pan, obspt_tilt, obspt_range,
									fgpts.points_pan[s_indx*LIDARBGSUB_MAX_BIN_PTS + j], fgpts.points_tilt[s_indx*LIDARBGSUB_MAX_BIN_PTS + j],
									fgpts.points_range[s_indx*LIDARBGSUB_MAX_BIN_PTS + j]) < epsilon)
								{
									//check parent
									my_parent = (core[(s_indx)*LIDARBGSUB_MAX_BIN_PTS + j] &&
										parentsin[(s_indx)*LIDARBGSUB_MAX_BIN_PTS + j] > my_parent) ?
										parentsin[(s_indx)*LIDARBGSUB_MAX_BIN_PTS + j] : my_parent;
								}
							}
						}
					}
				}
				parentsout[pind] = my_parent;
			}
		}
		//larger merges using atomics
		__global__ void PDSCAN_global(const LidarBin fgpts, const int * parentsin, int * parentsout, const bool *core, int rows, int cols, float epsilon, int searchwidth)
		{
			if (blockIdx.x >= cols || blockIdx.y >= rows) return;
			const int idx = (blockIdx.x + blockIdx.y*cols);
			const int pind = idx * LIDARBGSUB_MAX_BIN_PTS + threadIdx.x;
			if (threadIdx.x < fgpts.npts[idx])
			{
				//get your currecnt root
				int my_root = parentsin[pind];
				int it = 0;
				while (it < 10 && my_root >= 0 && my_root < rows*cols*LIDARBGSUB_MAX_BIN_PTS  && my_root != parentsin[my_root])
				{
					my_root = parentsin[my_root];
					it++;
				}

				float obspt_pan = fgpts.points_pan[pind];
				float obspt_tilt = fgpts.points_tilt[pind];
				float obspt_range = fgpts.points_range[pind];

				for (int ind_y = (int)(blockIdx.y) - 2; ind_y <= blockIdx.y + 2; ind_y++)
				{
					if (ind_y >= 0 && ind_y < rows)
					{
						for (int i_x = -searchwidth; i_x <= searchwidth; i_x++)
						{
							//wrapping
							int s_indx = ind_y * cols + ((i_x + blockIdx.x + cols) % cols);

							for (int j = 0; j < fgpts.npts[s_indx] && j < LIDARBGSUB_MAX_BIN_PTS; j++)
							{
								if (core[s_indx*LIDARBGSUB_MAX_BIN_PTS + j] && getDist(obspt_pan, obspt_tilt, obspt_range,
									fgpts.points_pan[s_indx*LIDARBGSUB_MAX_BIN_PTS + j], fgpts.points_tilt[s_indx*LIDARBGSUB_MAX_BIN_PTS + j],
									fgpts.points_range[s_indx*LIDARBGSUB_MAX_BIN_PTS + j]) < epsilon)
								{
									int otherroot = parentsin[s_indx*LIDARBGSUB_MAX_BIN_PTS + j];
									it = 0;
									while (it < 10 && otherroot >= 0 && otherroot < rows*cols && otherroot != parentsin[otherroot])
									{
										otherroot = parentsin[otherroot];
										it++;
									}

									if (otherroot > my_root)
									{
										atomicMax(&(parentsout[my_root < 0 ? pind : my_root]), otherroot);
										my_root = parentsout[my_root < 0 ? pind : my_root];
									}
								}
							}
						}
					}
				}
				atomicMax(&parentsout[pind], my_root);
			}

		}

		__global__ void setToRoot(const int * parentsin, int * parentsout, int npoints)
		{
			const int index = blockIdx.x * blockDim.x + threadIdx.x;
			if (index < npoints && parentsin[index] >= 0)
			{
				int head = parentsin[index];
				while (head >= 0 && head < npoints && head != parentsin[head])
					head = parentsin[head];
				parentsout[index] = head;
			}

		}


		std::vector<std::vector<DidoLidar_rangeData>> parseGraphsvec(LidarBin points, int * parents, int nbins)
		{
#ifdef _WIN32
			concurrency::concurrent_unordered_map<int, std::vector<DidoLidar_rangeData>> rootsinds;

			concurrency::parallel_for(0, nbins, [&](int i)
			{
				for (unsigned int j = 0; j < points.npts[i] && j < LIDARBGSUB_MAX_BIN_PTS; j++)
				{
					/*get it's root*/
					int root = parents[i*LIDARBGSUB_MAX_BIN_PTS + j];
					if (root < 0) continue;	//skip the special valued ones

					auto found = rootsinds.find(root);
					/*add it to that blob*/
					if (found != rootsinds.end())
					{
						found->second.push_back(DidoLidar_rangeData(points.points_range[i*LIDARBGSUB_MAX_BIN_PTS + j],
							points.points_pan[i*LIDARBGSUB_MAX_BIN_PTS + j], points.points_tilt[i*LIDARBGSUB_MAX_BIN_PTS + j]));
					}
					/*else create a new blob*/
					else
					{
						std::vector<DidoLidar_rangeData> temp;
						temp.push_back(DidoLidar_rangeData(points.points_range[i*LIDARBGSUB_MAX_BIN_PTS + j],
							points.points_pan[i*LIDARBGSUB_MAX_BIN_PTS + j], points.points_tilt[i*LIDARBGSUB_MAX_BIN_PTS + j]));
						rootsinds[root] = temp;
					}
				}

			}

			);
			//parse out the vectors
			std::vector<std::vector<DidoLidar_rangeData>> retval;
			for (auto & p : rootsinds)
			{
				retval.push_back(std::move(p.second));
			}
			return retval;

#else
			std::unordered_map<int, int> rootsinds;
			std::vector<std::vector<DidoLidar_rangeData>> retval;
			for (int i = 0; i < nbins; i++)
			{
				for (unsigned int j = 0; j < points.npts[i] && j < LIDARBGSUB_MAX_BIN_PTS; j++)
				{
					/*get it's root*/
					int root = parents[i*LIDARBGSUB_MAX_BIN_PTS + j];
					if (root < 0) continue;	//skip the special valued ones

					auto found = rootsinds.find(root);
					/*add it to that blob*/
					if (found != rootsinds.end())
					{
						retval[found->second].push_back(DidoLidar_rangeData(points.points_range[i*LIDARBGSUB_MAX_BIN_PTS + j],
							points.points_pan[i*LIDARBGSUB_MAX_BIN_PTS + j], points.points_tilt[i*LIDARBGSUB_MAX_BIN_PTS + j]));
					}
					/*else create a new blob*/
					else
					{
						rootsinds[root] = retval.size();
						std::vector<DidoLidar_rangeData> temp;
						temp.push_back(DidoLidar_rangeData(points.points_range[i*LIDARBGSUB_MAX_BIN_PTS + j],
							points.points_pan[i*LIDARBGSUB_MAX_BIN_PTS + j], points.points_tilt[i*LIDARBGSUB_MAX_BIN_PTS + j]));
						retval.push_back(temp);
					}
				}
			}
			return retval;
#endif
		}


	}

	DidoAnalytics_LidarBGSUB::DidoAnalytics_LidarBGSUB(float binw, float binh, float mindist, int history, float bgthreshold, float _eps, int _ncore) :
		binWidth(binw), binHeight(binh), sqmindist(mindist*mindist), hist(history), bgthresh(bgthreshold), epsilon(_eps), ncore(_ncore)
	{
		modelCols = abs((int)std::floor(2 * HIP_PI_F / binw)) + 2;
		//no lidar in production I know of has a verticual FOV greater than 90' - maybe this should be a parameter?
		modelRows = abs((int)std::floor(HIP_PIO2_F / binh)) + 2;
		searchWidth = (int)(mindist / binw) + 1;
		if (modelCols*modelRows < 1) throw std::runtime_error("the model size must be at least one in both dimensions");
#if DIDOLIDAR_NOGPU

#else
		bgmodel.allocate(modelCols*modelRows);
		HANDLE_ERROR(hipMemset(bgmodel.npts, 0, modelCols*modelRows * sizeof(unsigned int)));
		HANDLE_ERROR(hipMalloc(&modelWeights, modelCols*modelRows * sizeof(float)*LIDARBGSUB_MAX_BIN_PTS));
		HANDLE_ERROR(hipMemset(modelWeights, 0, modelCols*modelRows * sizeof(float)*LIDARBGSUB_MAX_BIN_PTS));
		HANDLE_ERROR(hipMalloc(&variances, modelCols*modelRows * sizeof(float)*LIDARBGSUB_MAX_BIN_PTS));
		HANDLE_ERROR(hipMemset(variances, 0, modelCols*modelRows * sizeof(float)*LIDARBGSUB_MAX_BIN_PTS));
#endif
	}


	DidoAnalytics_LidarBGSUB::~DidoAnalytics_LidarBGSUB()
	{
#if DIDOLIDAR_NOGPU

#else
		bgmodel.deallocate();
		safe_Free(modelWeights);
		safe_Free(variances);
#endif
	}



	std::vector<DidoLidar_rangeData> DidoAnalytics_LidarBGSUB::apply(const DidoLidar_rangeData* points, int npts, float learningRate)
	{
#if DIDOLIDAR_NOGPU
		std::vector<DidoLidar_rangeData> rval;
#else
		frameno++;
		float lr = (learningRate < 0) ? 1.0f / min(frameno, hist) : learningRate;

		if (lr != lr) throw std::runtime_error("learning rate was NaN");

#if DEBUG_TIMINGS
		auto prevtime = std::chrono::high_resolution_clock::now();
#endif

		//create GPU allocated data
		LidarBin d_obs;
		DidoLidar_rangeData * d_pts;
		int * d_nout;
		bool * d_addToModel;
		if (modelCols*modelRows < 1) throw std::runtime_error("the model size must be at least one in both dimensions");

		d_obs.allocate(modelCols*modelRows);
		HANDLE_ERROR(hipMemset(d_obs.npts, 0, modelCols*modelRows * sizeof(unsigned int))); //make sure it's at zero
		HANDLE_ERROR(hipMalloc(&d_pts, npts * sizeof(DidoLidar_rangeData)));
		HANDLE_ERROR(hipMalloc(&d_nout, sizeof(int)));
		HANDLE_ERROR(hipMalloc(&d_addToModel, modelCols*modelRows * sizeof(bool) * LIDARBGSUB_MAX_BIN_PTS));
		HANDLE_ERROR(hipMemset(d_addToModel, 0, modelCols*modelRows * sizeof(bool) * LIDARBGSUB_MAX_BIN_PTS)); //make sure it's at zero
		HANDLE_ERROR(hipMemcpy(d_pts, points, npts * sizeof(DidoLidar_rangeData), hipMemcpyHostToDevice));

#if DEBUG_TIMINGS
		auto ts = std::chrono::high_resolution_clock::now();
		std::cout << "allocation took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		lbgsCUDA::collatePoints << <npts / 128 + 1, 128 >> >(d_obs, d_pts, npts, modelRows, modelCols, binWidth, binHeight);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());
		//run the bgsub
		dim3 grid(modelCols, modelRows);
		dim3 block(LIDARBGSUB_MAX_BIN_PTS, 1);


#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "colaltion took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		lbgsCUDA::bgsubKernel_pt1 << <grid, block >> >(bgmodel, d_obs, variances, modelWeights, bgthresh, sqmindist, modelRows, modelCols, lr, searchWidth);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());


#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "subtraction part 1 took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif

		lbgsCUDA::bgsubKernel_pt2 << <grid, block >> >(bgmodel, d_obs, d_pts, d_nout, variances, modelWeights, d_addToModel, bgthresh, modelRows, modelCols, searchWidth);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());


#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "subtraction pt2 took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		//parse the data to the output
		lbgsCUDA::sortModels << <grid, block >> >(bgmodel, d_obs, d_addToModel, variances, modelWeights, modelRows, modelCols, lr, sqmindist*0.75f);
		int * nout = (int*)malloc(sizeof(int));
		HANDLE_ERROR(hipMemcpy(nout, d_nout, sizeof(int), hipMemcpyDeviceToHost));
		std::vector<DidoLidar_rangeData> rval(nout[0]);
		HANDLE_ERROR(hipMemcpy(rval.data(), d_pts, nout[0] * sizeof(DidoLidar_rangeData), hipMemcpyDeviceToHost));
		if (nout) free(nout);
		hipDeviceSynchronize();

#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "sorting took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		HANDLE_ERROR(hipGetLastError());
		safe_Free(d_addToModel);
		d_obs.deallocate();
		safe_Free(d_pts);
		safe_Free(d_nout);
#endif
		return rval;
	}



	std::vector<std::vector<DidoLidar_rangeData>> DidoAnalytics_LidarBGSUB::applyAndCluster(const DidoLidar_rangeData * points, int npts, float learningRate)
	{
#if DIDOLIDAR_NOGPU
		std::vector<std::vector<DidoLidar_rangeData>> rval;
#else
		frameno++;
		float lr = (learningRate < 0) ? 1.0f / min(frameno, hist) : learningRate;

		if (lr != lr) throw std::runtime_error("learning rate was NaN");


#if DEBUG_TIMINGS
		auto prevtime = std::chrono::high_resolution_clock::now();
#endif

		//create GPU allocated data
		LidarBin d_obs;
		DidoLidar_rangeData * d_pts;
		int * d_nout;
		bool * d_addToModel;

		//clustering data
		LidarBin d_cluster_obs;
		int * d_cluster_parents_1, *d_cluster_parents_2;
		bool * d_cluster_core;

		float clusterWidth = min(binWidth * 4, epsilon / 3);
		float clusterHeight = min(binHeight * 4, epsilon / 3);
		int clusterCols = (int)std::floor(2 * HIP_PI_F / clusterWidth) + 2;
		int clusterRows = (int)std::floor(binHeight*modelRows / clusterHeight) + 1;
		int clusterSearch = (int)(epsilon / clusterWidth) + 1;

		if (clusterCols*clusterRows < 1) throw std::runtime_error("the cluster size must be at least one in both dimensions");
		if (modelCols*modelRows < 1) throw std::runtime_error("the model size must be at least one in both dimensions");

		d_cluster_obs.allocate(clusterCols*clusterRows);
		HANDLE_ERROR(hipMemset(d_cluster_obs.npts, 0, clusterCols*clusterRows * sizeof(unsigned int))); //make sure it's at zero
		HANDLE_ERROR(hipMalloc(&d_cluster_core, clusterCols*clusterRows * sizeof(bool) * LIDARBGSUB_MAX_BIN_PTS));
		HANDLE_ERROR(hipMalloc(&d_cluster_parents_1, clusterCols*clusterRows * sizeof(int) * LIDARBGSUB_MAX_BIN_PTS));
		HANDLE_ERROR(hipMalloc(&d_cluster_parents_2, clusterCols*clusterRows * sizeof(int) * LIDARBGSUB_MAX_BIN_PTS));

		d_obs.allocate(modelRows*modelCols);
		HANDLE_ERROR(hipMemset(d_obs.npts, 0, modelCols*modelRows * sizeof(unsigned int))); //make sure it's at zero
		HANDLE_ERROR(hipMalloc(&d_pts, npts * sizeof(DidoLidar_rangeData)));
		HANDLE_ERROR(hipMalloc(&d_nout, sizeof(int)));
		HANDLE_ERROR(hipMalloc(&d_addToModel, modelCols*modelRows * sizeof(bool) * LIDARBGSUB_MAX_BIN_PTS));
		HANDLE_ERROR(hipMemset(d_addToModel, 0, modelCols*modelRows * sizeof(bool) * LIDARBGSUB_MAX_BIN_PTS)); //make sure it's at zero
		HANDLE_ERROR(hipMemcpy(d_pts, points, npts * sizeof(DidoLidar_rangeData), hipMemcpyHostToDevice));


#if DEBUG_TIMINGS
		auto ts = std::chrono::high_resolution_clock::now();
		std::cout << "allocation took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		lbgsCUDA::collatePoints << <npts / 128 + 1, 128 >> >(d_obs, d_pts, npts, modelRows, modelCols, binWidth, binHeight);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());
		//run the bgsub
		dim3 grid(modelCols, modelRows);
		dim3 block(LIDARBGSUB_MAX_BIN_PTS, 1);


#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "colaltion took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		lbgsCUDA::bgsubKernel_pt1 << <grid, block >> >(bgmodel, d_obs, variances, modelWeights, bgthresh, sqmindist, modelRows, modelCols, lr, searchWidth);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());

#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "subtraction part 1 took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		lbgsCUDA::bgsubKernel_pt2 << <grid, block >> >(bgmodel, d_obs, d_pts, d_nout, variances, modelWeights, d_addToModel, bgthresh, modelRows, modelCols, searchWidth);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());


#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "bgsub pt 2 took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		//parse the data to the output
		lbgsCUDA::sortModels << <grid, block >> >(bgmodel, d_obs, d_addToModel, variances, modelWeights, modelRows, modelCols, lr, sqmindist*0.75f);

		int * nout = (int*)malloc(sizeof(int));
		HANDLE_ERROR(hipMemcpy(nout, d_nout, sizeof(int), hipMemcpyDeviceToHost));

		//collate the foreground points for clustering
		lbgsCUDA::collatePoints << <nout[0] / 128 + 1, 128 >> >(d_cluster_obs, d_pts, nout[0], clusterRows, clusterCols, clusterWidth, clusterHeight);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());


#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "sorting and clustering collation took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif

		dim3 clustergrid(clusterCols, clusterRows);
		//initialise the parents
		lbgsCUDA::PDSCAN_init << <clustergrid, block >> >(d_cluster_obs, d_cluster_parents_1, d_cluster_core, clusterRows, clusterCols, epsilon, ncore, clusterSearch);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());

#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "pdbscan init took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif
		//quickly do local updates
		lbgsCUDA::PDSCAN_local << <clustergrid, block >> >(d_cluster_obs, d_cluster_parents_1, d_cluster_parents_2, d_cluster_core, clusterRows, clusterCols, epsilon, clusterSearch);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());

#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "pdbscan local took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif

		//now update the roots
		lbgsCUDA::PDSCAN_global << <clustergrid, block >> >(d_cluster_obs, d_cluster_parents_2, d_cluster_parents_1, d_cluster_core, clusterRows, clusterCols, epsilon, clusterSearch);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());
		//twice so that we get everything
		lbgsCUDA::PDSCAN_global << <clustergrid, block >> >(d_cluster_obs, d_cluster_parents_1, d_cluster_parents_2, d_cluster_core, clusterRows, clusterCols, epsilon, clusterSearch);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());


#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "pdbscan global took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		lbgsCUDA::setToRoot << <clusterRows*clusterCols, LIDARBGSUB_MAX_BIN_PTS >> >(d_cluster_parents_2, d_cluster_parents_1, clusterRows*clusterCols*LIDARBGSUB_MAX_BIN_PTS);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());


#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "setting to root took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif

		//now drop it onto the cpu and compute the output
		LidarBin lbins;
		lbins.local_allocate(clusterRows*clusterCols);
		std::vector<int> lparents(clusterRows*clusterCols*LIDARBGSUB_MAX_BIN_PTS);

		HANDLE_ERROR(hipMemcpy(lparents.data(), d_cluster_parents_1, clusterRows*clusterCols*LIDARBGSUB_MAX_BIN_PTS * sizeof(int), hipMemcpyDeviceToHost));
		lbins.copyDownFrom(d_cluster_obs, clusterRows*clusterCols);

		auto rval = lbgsCUDA::parseGraphsvec(lbins, lparents.data(), clusterRows*clusterCols);
		lbins.local_deallocate();
		if (nout) free(nout);
		hipDeviceSynchronize();


#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "generating output took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		HANDLE_ERROR(hipGetLastError());
		safe_Free(d_cluster_parents_1);
		safe_Free(d_cluster_parents_2);
		d_cluster_obs.deallocate();
		safe_Free(d_cluster_core);
		safe_Free(d_addToModel);
		d_obs.deallocate();
		safe_Free(d_pts);
		safe_Free(d_nout);
#endif
		return rval;
	}

	std::vector<std::vector<DidoLidar_rangeData>> DidoAnalytics_LidarBGSUB::Cluster(const DidoLidar_rangeData * fgpts, size_t npts)
	{
#if DIDOLIDAR_NOGPU
		std::vector<std::vector<DidoLidar_rangeData>> rval;
#else
		//create GPU allocated data
		DidoLidar_rangeData * d_pts;

		//clustering data
		LidarBin d_cluster_obs;
		int * d_cluster_parents_1, *d_cluster_parents_2;
		bool * d_cluster_core;

		float clusterWidth = min(binWidth * 4, epsilon / 3);
		float clusterHeight = min(binHeight * 4, epsilon / 3);
		int clusterCols = (int)std::floor(2 * HIP_PI_F / clusterWidth) + 2;
		int clusterRows = (int)std::floor(binHeight*modelRows / clusterHeight) + 1;
		int clusterSearch = (int)(epsilon / clusterWidth) + 1;

		if (clusterCols*clusterRows < 1) throw std::runtime_error("the cluster size must be at least one in both dimensions");


#if DEBUG_TIMINGS
		auto prevtime = std::chrono::high_resolution_clock::now();
#endif

		d_cluster_obs.allocate(clusterCols*clusterRows * sizeof(unsigned int));
		HANDLE_ERROR(hipMemset(d_cluster_obs.npts, 0, clusterCols*clusterRows * sizeof(unsigned int))); //make sure it's at zero
		HANDLE_ERROR(hipMalloc(&d_cluster_core, clusterCols*clusterRows * sizeof(bool) * LIDARBGSUB_MAX_BIN_PTS));
		HANDLE_ERROR(hipMalloc(&d_cluster_parents_1, clusterCols*clusterRows * sizeof(int) * LIDARBGSUB_MAX_BIN_PTS));
		HANDLE_ERROR(hipMalloc(&d_cluster_parents_2, clusterCols*clusterRows * sizeof(int) * LIDARBGSUB_MAX_BIN_PTS));

		HANDLE_ERROR(hipMalloc(&d_pts, npts * sizeof(DidoLidar_rangeData)));
		HANDLE_ERROR(hipMemcpy(d_pts, fgpts, npts * sizeof(DidoLidar_rangeData), hipMemcpyHostToDevice));


#if DEBUG_TIMINGS
		auto ts = std::chrono::high_resolution_clock::now();
		std::cout << "cluster allocation took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		//collate the foreground points for clustering
		lbgsCUDA::collatePoints << <(npts / 128) + 1, 128 >> >(d_cluster_obs, d_pts, npts, clusterRows, clusterCols, clusterWidth, clusterHeight);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());

#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "cluster collation took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		dim3 block(LIDARBGSUB_MAX_BIN_PTS, 1);
		dim3 clustergrid(clusterCols, clusterRows);
		//initialise the parents
		lbgsCUDA::PDSCAN_init << <clustergrid, block >> >(d_cluster_obs, d_cluster_parents_1, d_cluster_core, clusterRows, clusterCols, epsilon, ncore, clusterSearch);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());

#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "pdbscan inti took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif


		//quickly do local updates
		lbgsCUDA::PDSCAN_local << <clustergrid, block >> >(d_cluster_obs, d_cluster_parents_1, d_cluster_parents_2, d_cluster_core, clusterRows, clusterCols, epsilon, clusterSearch);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());

#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "pdbscan local took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif
		//now update the roots
		lbgsCUDA::PDSCAN_global << <clustergrid, block >> >(d_cluster_obs, d_cluster_parents_2, d_cluster_parents_1, d_cluster_core, clusterRows, clusterCols, epsilon, clusterSearch);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());
		//twice so that we get everything
		lbgsCUDA::PDSCAN_global << <clustergrid, block >> >(d_cluster_obs, d_cluster_parents_1, d_cluster_parents_2, d_cluster_core, clusterRows, clusterCols, epsilon, clusterSearch);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());

#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "pdbscan global took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif

		lbgsCUDA::setToRoot << <clusterRows*clusterCols, LIDARBGSUB_MAX_BIN_PTS >> >(d_cluster_parents_2, d_cluster_parents_1, clusterRows*clusterCols*LIDARBGSUB_MAX_BIN_PTS);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());


#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "set to root took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif

		//now drop it onto the cpu and compute the output
		LidarBin lbins;
		lbins.local_allocate(clusterRows*clusterCols);
		std::vector<int> lparents(clusterRows*clusterCols*LIDARBGSUB_MAX_BIN_PTS);

		HANDLE_ERROR(hipMemcpy(lparents.data(), d_cluster_parents_1, clusterRows*clusterCols*LIDARBGSUB_MAX_BIN_PTS * sizeof(int), hipMemcpyDeviceToHost));
		lbins.copyDownFrom(d_cluster_obs, clusterRows*clusterCols);

		auto rval = lbgsCUDA::parseGraphsvec(lbins, lparents.data(), clusterRows*clusterCols);
		hipDeviceSynchronize();


#if DEBUG_TIMINGS
		ts = std::chrono::high_resolution_clock::now();
		std::cout << "generating output took " << std::chrono::duration_cast<std::chrono::milliseconds>(ts - prevtime).count() << "ms" << std::endl;
		prevtime = ts;
#endif

		lbins.local_deallocate();
		HANDLE_ERROR(hipGetLastError());
		safe_Free(d_cluster_parents_1);
		safe_Free(d_cluster_parents_2);
		d_cluster_obs.deallocate();
		safe_Free(d_cluster_core);
		safe_Free(d_pts);
#endif
		return rval;
	}




	__global__ void countPoints(LidarBin bins, unsigned char* out, int npts)
	{
		const int index = threadIdx.x + blockIdx.x * blockDim.x;
		if (index < npts)
		{
			out[index] = (unsigned char)bins.npts[index];
		}
	}


	void DidoAnalytics_LidarBGSUB::dispayBgmodelNpts(unsigned char * out, int npts)
	{
		unsigned char * d_counts;
		HANDLE_ERROR(hipMalloc(&d_counts, modelRows*modelCols * sizeof(unsigned char)));
		countPoints << <modelRows*modelCols / 128 + 1, 128 >> > (bgmodel, d_counts, modelRows*modelCols);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());
		HANDLE_ERROR(hipMemcpy(out, d_counts, min(modelRows*modelCols, npts) * sizeof(unsigned char), hipMemcpyDeviceToHost));
		safe_Free(d_counts);
	}

	void LidarBin::allocate(size_t nbins)
	{
		HANDLE_ERROR(hipMalloc(&npts, nbins * sizeof(unsigned int)));
		HANDLE_ERROR(hipMalloc(&points_pan, nbins*LIDARBGSUB_MAX_BIN_PTS * sizeof(float)));
		HANDLE_ERROR(hipMalloc(&points_tilt, nbins*LIDARBGSUB_MAX_BIN_PTS * sizeof(float)));
		HANDLE_ERROR(hipMalloc(&points_range, nbins*LIDARBGSUB_MAX_BIN_PTS * sizeof(float)));
	}
	void LidarBin::local_allocate(size_t nbins)
	{
		npts = (unsigned int *)malloc(nbins * sizeof(unsigned int));
		points_pan = (float*)malloc(nbins*LIDARBGSUB_MAX_BIN_PTS * sizeof(float));
		points_tilt = (float*)malloc(nbins*LIDARBGSUB_MAX_BIN_PTS * sizeof(float));
		points_range = (float*)malloc(nbins*LIDARBGSUB_MAX_BIN_PTS * sizeof(float));
	}
	void LidarBin::deallocate()
	{
		safe_Free(npts);
		safe_Free(points_pan);
		safe_Free(points_tilt);
		safe_Free(points_range);
	}
	void LidarBin::local_deallocate()
	{
		if (npts) free(npts);
		if (points_pan) free(points_pan);
		if (points_pan) free(points_tilt);
		if (points_pan) free(points_range);

	}
	void LidarBin::copyDownFrom(LidarBin & src, size_t nbins)
	{
		HANDLE_ERROR(hipMemcpy(npts, src.npts, nbins * sizeof(unsigned int), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(points_pan, src.points_pan, nbins*LIDARBGSUB_MAX_BIN_PTS * sizeof(float), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(points_tilt, src.points_tilt, nbins*LIDARBGSUB_MAX_BIN_PTS * sizeof(float), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(points_range, src.points_range, nbins*LIDARBGSUB_MAX_BIN_PTS * sizeof(float), hipMemcpyDeviceToHost));
	}
}