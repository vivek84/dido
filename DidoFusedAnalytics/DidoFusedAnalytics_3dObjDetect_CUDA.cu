#include "hip/hip_runtime.h"
/**  *****************************************************************************
* This program is the confidential and proprietary product of Overview    
* Limited. Any unauthorised use, reproduction or transfer of this         
* program is strictly prohibited.              
* Copyright 2017 Overview Limited. (Subject to limited                    
* distribution and restricted disclosure only.) All rights reserved.
*
* @file    	DidoFusedAnalytics_3dObjDetect_CUDA.h
* @author  	SL
* @version 	1
* @date    	2017-07-05
* @brief   	class that does the blob detection on the GPU
 *****************************************************************************
**/



/**
*	description of the algorithm
*	collects the objects using PDSScan. This method searches the neighbourhood around each point and labels the point as core if it has above a threshold of neighbours
*	then points that are neighbours with core points are joined to the same cluster using a tree structure. This is done in parallel by repeated steps of searching the neighbourhood
*	and taking the lowest parent from amongst the parents of the core points in your neighbourhood until all the points in the block have collected their heads.
*	then each point is labelled with the root of it's tree. These points are then sorted by their parent index using merge sort. Finally this sorted list of points is formed into detections
*	in a two stage process, where first blocks of points are combined into sets of bounding boxes before this list of bounding boxes is then further combined on the CPU in the final pass to give the output

*/

#include "global_defines.h"
#include <vector>
#include "DidoFusedAnalytics_3dObjDetect_CUDA.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include "math.h"
#include "hip/hip_math_constants.h"
#include "hip/hip_runtime_api.h"
#include <chrono>


//for our logging we will throw an error that can then be caught by the surrounding code that is allowed to include boost
#include "CUDA_Exception.h"


/*
   if the computer being used doesn't have a GPU, define DIDOLIDAR_NOGPU as 1 in the preprocessor, and this wil produce some noops instead. It still requires the nvidia sdk to compile at all, however
*/



#if DIDOLIDAR_NOGPU

#else
//error handling function
static void HandleError( hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
	//	hipDeviceReset();
     throw overview::CUDA_Exception(hipGetErrorString( err ) , err, line, file);
    }
}
#define HANDLE_ERROR(err) {HandleError((err), __FILE__, __LINE__);} 
#endif

namespace overview
{

namespace objdetectCUDA
{
#if DIDOLIDAR_NOGPU

#else

//datatype for handling our things
	//prefer box of pointers to pointer to boxes in cuda
struct node
{
	bool * valid;
	bool * core;
	int * parentind;
	int * index;
};

//populates a box of nodes stored row major
__global__ void generateTrees(const float * fg_ranges_min, const float * fg_ranges_max, node outnodes, int rows, int cols)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if (x < cols && y < rows)
	{
		//work out if you are a node
		bool cond = (fg_ranges_max[x + cols*y] > 0 && fg_ranges_min[x + cols*y] > 0);
		outnodes.valid[x + cols*y] = cond;
		outnodes.parentind[x + cols*y] = cond ? x + cols*y : -1;
		outnodes.index[x + cols*y] = x + cols*y;
	}
}

//function to calculate the separation between ranges of ranges
__device__  inline float rangeDist(const float & a_min, const float & a_max, const float & b_min, const float & b_max)
{
	//look at the separation of thje averages

	//check if any of the points are at infinity (meaninbg there was no range observation)
	if(isfinite(a_min) && isfinite(a_max) && isfinite(b_min) && isfinite(b_max))
		return (abs((a_min + a_max) / 2 - (b_min + b_max) / 2) );
	else return 0;
}


//parallel DBSCAN using propagation
//this version wworks on a shred copy, but doens't seem to get every point (maybe we have to call it twice)
template <int eps, int bDim>
__global__ void PDSDBSCANInit__shared(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	const int l_len = (bDim + 2 * eps);
	const int l_size = l_len*l_len;
	__shared__ __align__(8) int l_parents[l_size];
	__shared__ __align__(8) float l_ranges_min[l_size];
	__shared__ __align__(8) float l_ranges_max[l_size];
	__shared__ __align__(4) bool l_valid[l_size];
	__shared__ __align__(4) bool l_core[l_size];

	//populate the local storage
	int flatind = threadIdx.y + threadIdx.x*blockDim.y;
	if (flatind < l_size / 2)
	{
		int xind = (cols + blockIdx.x * blockDim.x - eps + (flatind % l_len))%cols;
		int yind = blockIdx.y * blockDim.y - eps + (flatind / l_len);
		bool cond = yind >= 0 && yind < rows;
		l_ranges_min[flatind] = cond ? fg_ranges_min[yind*cols + xind] : -1.0f;
		l_ranges_max[flatind] = cond ? fg_ranges_max[yind*cols + xind] : -1.0f;
		l_valid[flatind] = cond ? nodes.valid[yind*cols + xind] : false;
		l_parents[flatind] = cond ? nodes.parentind[yind*cols + xind] : -1;
		l_core[flatind] = cond ? nodes.core[yind*cols + xind] : false;
		xind = (blockIdx.x * blockDim.x - eps + ((flatind + (l_size / 2)) % l_len));
		yind = blockIdx.y * blockDim.y - eps + ((flatind + (l_size / 2) )/ l_len);
		cond = yind >= 0 && yind < rows && xind >= 0 && xind < cols;
		l_ranges_min[flatind + (l_size / 2)] = cond ? fg_ranges_min[yind*cols + xind] : -1.0f;
		l_ranges_max[flatind + (l_size / 2)] = cond ? fg_ranges_max[yind*cols + xind] : -1.0f;
		l_valid[flatind + (l_size / 2)] = cond ? nodes.valid[yind*cols + xind] : false;
		l_parents[flatind + (l_size / 2)] = cond ? nodes.parentind[yind*cols + xind] : -1;
		l_core[flatind + (l_size / 2)] = cond ? nodes.core[yind*cols + xind] : false;
	}

	int l_ind = l_len*(threadIdx.y + eps) + (threadIdx.x + eps);
	if (x >= cols || y >= rows) return;
	if (l_valid[l_ind])
	{
		//check every point in your region to decide if you are a core point and populate your neigbourhood vector
		int nneighbours = 0;
		float myrange_min = l_ranges_min[l_ind];
		float myrange_max = l_ranges_max[l_ind];

		for (int i = -eps; i <= eps; i++)
		{
			int xind = eps + threadIdx.x + i;
			//manhattan distances are cheaper to compute
			for (int j = abs(i) - eps; j <= eps - abs(i); j++)
			{
				int t_lind = xind + l_len*(eps + threadIdx.y + j);
				//check range
				float sep = rangescaling*rangeDist(l_ranges_min[t_lind], l_ranges_max[t_lind],  myrange_min, myrange_max) + abs(i) + abs(j);
				nneighbours += l_valid[t_lind] && (sep <= eps);
			}
		}
		l_core[l_ind] = (nneighbours >= ncore);
		l_parents[l_ind] = l_core[l_ind] ? l_parents[l_ind] : -1;
		__syncthreads();

		__align__(4) int my_parent = l_parents[l_ind];
		for (int s = 0; s < bDim / eps; s++)
		{
			for (int i = -eps; i <= eps; i++)
			{
				int xind = eps + threadIdx.x + i;
				for (int j = abs(i) - eps; j <= eps - abs(i); j++)
				{
					int t_lind = xind + l_len*(eps + threadIdx.y + j);
					float sep = rangescaling*rangeDist(l_ranges_min[t_lind], l_ranges_max[t_lind], myrange_min, myrange_max) + abs(i) + abs(j);
					if ((sep <= eps) && l_valid[t_lind])
						my_parent = (l_core[t_lind] && (l_parents[t_lind] > my_parent)) ? l_parents[t_lind] : my_parent;
				}
			}
			l_parents[l_ind] = my_parent;
			__syncthreads();
		}
	}
	//drop values back to global
	nodes.core[x + y*cols] = l_core[l_ind];
	nodes.parentind[x + y*cols] = l_parents[l_ind];
} 

template <int eps>
__global__ void PDSDBSCANLocal(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore)
{
	const int maxnbour = eps * 2 * (eps + 1) + 1;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (!(x < cols && y < rows && nodes.valid[x + y*cols])) return;
	//check every point in your region to populate your neigbourhood vector
	__align__(4) bool nbhood[maxnbour];
	int nneighbours = 0;
	float myrange_min = fg_ranges_min[x + y*cols];
	float myrange_max = fg_ranges_max[x + y*cols];

	int my_parent = nodes.core[x + cols*y] ? nodes.parentind[x + cols*y] : -1;
	for (int i = -eps; i <= eps && nneighbours < maxnbour; i++)
	{
		int xind = (cols + x + i) % cols;
		//manhattan distances are cheaper to compute
		for (int j = abs(i) - eps; j <= eps - abs(i) && nneighbours < maxnbour; j++)
		{
			int yind = y + j;
			//bounds check
			if (yind >= 0 && yind < rows && nodes.valid[xind + cols*yind])
			{
				//check range
				float range = rangescaling*rangeDist(fg_ranges_min[xind + cols*yind], fg_ranges_max[xind + cols*yind], myrange_min, myrange_max) + abs(i) + abs(j);
				nbhood[nneighbours] = range <= eps;
				//check parent
				my_parent = (nbhood[nneighbours] && nodes.core[xind + cols*yind] &&
					(nodes.parentind[xind + cols*yind] > my_parent)) ? 
					  nodes.parentind[xind + cols*yind] : my_parent;
			}
			else
			{
				nbhood[nneighbours] = false;
			}
			nneighbours++;
		}
	}
	nodes.parentind[x + cols*y] = my_parent;
	__syncthreads();

	//get your currecnt root
	int my_root = my_parent;
	int it = 0;
	while ( it < 10 && my_root >=0 && my_root < rows*cols  && my_root != nodes.parentind[my_root])
	{
			my_root = nodes.parentind[my_root];
			it++;
	}

	nneighbours = 0;
	for (int i = -eps; i <= eps && nneighbours < maxnbour; i++)
	{
		int xind = (cols + x + i) % cols;
		//manhattan distances are cheaper to compute
		for (int j = abs(i) - eps; j <= eps - abs(i) && nneighbours < maxnbour; j++)
		{
			int yind = y + j;
			if (nbhood[nneighbours] && nodes.core[xind + cols*yind] && nodes.parentind[xind + cols*yind] != my_parent)
			{
				int otherroot = nodes.parentind[xind + cols*yind];
				it = 0;
				while (it < 10 && otherroot >= 0 && otherroot < rows*cols && otherroot != nodes.parentind[otherroot])
				{
					otherroot = nodes.parentind[otherroot];
					it++;
				}

				if (otherroot > my_root)
				{
					atomicMax(&(nodes.parentind[my_root < 0 ? x + y*cols : my_root]), otherroot);
					my_root = nodes.parentind[my_root < 0 ? x + y*cols : my_root];
				}
			}
			nneighbours++;
		}
	}
}

template <int eps>
__global__ void PDSDBSCANInit(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore)
{
	static const int maxnbour = eps * 2 * (eps + 1) + 1;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (!(x < cols && y < rows && nodes.valid[x + y*cols])) return;
	//check every point in your region to decide if you are a core point and populate your neigbourhood vector
	int nneighbours = 0;
	float myrange_min = fg_ranges_min[x + y*cols];
	float myrange_max = fg_ranges_max[x + y*cols];

	for (int i = -eps; i <= eps && nneighbours < maxnbour; i++)
	{
		int xind = (x + i) ;
		//manhattan distances are cheaper to compute
		for (int j = abs(i) - eps; j <= eps - abs(i) && nneighbours < maxnbour; j++)
		{
			int yind = y + j;
			//bounds check
			if (yind >= 0 && yind < rows && xind >= 0 && xind < cols && nodes.valid[xind + cols*yind])
			{
				//check range
				float range = rangescaling*rangeDist(fg_ranges_min[xind + cols*yind], fg_ranges_max[xind + cols*yind], myrange_min, myrange_max) + abs(i) + abs(j);
				if (range <= eps)
				{
					nneighbours++;
				}
			}
		}
	}
	nodes.core[x + y*cols] = (nneighbours >= ncore);
}

//template initialisation
template __global__ void PDSDBSCANLocal<5>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);

template __global__ void PDSDBSCANInit__shared<5, 32>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);
template __global__ void PDSDBSCANInit<5>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);

template __global__ void PDSDBSCANLocal<4>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);

template __global__ void PDSDBSCANInit__shared<4, 32>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);
template __global__ void PDSDBSCANInit<4>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);

template __global__ void PDSDBSCANLocal<3>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);

template __global__ void PDSDBSCANInit__shared<3, 32>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);
template __global__ void PDSDBSCANInit<3>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);

template __global__ void PDSDBSCANLocal<2>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);

template __global__ void PDSDBSCANInit__shared<2, 32>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);
template __global__ void PDSDBSCANInit<2>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);

template __global__ void PDSDBSCANLocal<1>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);

template __global__ void PDSDBSCANInit__shared<1, 32>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);
template __global__ void PDSDBSCANInit<1>(const float * fg_ranges_min, const float * fg_ranges_max, node nodes, int rows, int cols, float rangescaling, int ncore);

__global__ void g_setToHead(const node nodes, int * temphead, int rowxcols, int nnodes)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < nnodes)
	{
		int head = (index < rowxcols) ? nodes.parentind[index] : -1;
		while (head >= 0 && head < nnodes && head != nodes.parentind[head])
			head = nodes.parentind[head];
		temphead[index] = head;
	}
}

__global__ void  applysort(node arr, int* ptrs, int * parents, int nnodes)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < nnodes)
	{
		arr.parentind[index] = parents[index];
		arr.index[index] = ptrs[index];
	}
}


const int nOEMthreads = 1024;
const int nOEMshare = nOEMthreads * 2;

__global__ void OEMSort_kernel(int* arr, int * g_parents, int nnodes)
{
	//we iterate up until the len is our kernelsize, so we can use local (faster) synchronises
	const int idx = blockIdx.x * nOEMshare + threadIdx.x;
	if (threadIdx.x < nOEMthreads && idx < nnodes)
	{
		//cache into shared memory
		__shared__ __align__(8) int cache[nOEMshare];
		__shared__ __align__(8) int parents[nOEMshare];

		parents[threadIdx.x] = g_parents[idx];
		cache[threadIdx.x] = idx;
		parents[threadIdx.x + nOEMthreads] = idx + nOEMthreads < nnodes ? g_parents[idx + nOEMthreads] : -1 ;
		cache[threadIdx.x + nOEMthreads] = idx + nOEMthreads;
		__syncthreads();
		for (int len = 2; len <= blockDim.x; len *= 2)
		{
			int section_idx = threadIdx.x / (len / 2); //there are len/2 comparisons in each block
			int step_idx = threadIdx.x % (len / 2);
			int i = step_idx + section_idx*len;
			int stepped = i + (len / 2);
			if (stepped < nOEMshare)
			{
				//compareAndExchangeNodes(cache + i, cache + i + (len / 2));
				bool comp = parents[i] < parents[stepped];
				int tmp = cache[comp ? stepped : i];
				int p = parents[comp ? stepped : i];
				parents[stepped] = parents[comp ? i : stepped];
				parents[i] = p;
				cache[stepped] = cache[comp ? i : stepped];
				cache[i] = tmp;
			}
			__syncthreads();
			for (int step = len / 4; step > 0; step /= 2)
			{
				int start_idx = step_idx % step;
				int it_idx = step_idx / step;

				i = section_idx*len + start_idx + step + it_idx * 2 * step;
				stepped = i + step;
				if ((step * 2 + it_idx * 2 * step < len) && (stepped < nOEMshare))
				{
					bool comp = parents[i] < parents[stepped];
					int tmp = cache[comp ? stepped : i];
					int p = parents[comp ? stepped : i];
					parents[stepped] = parents[comp ? i : stepped];
					parents[i] = p;
					cache[stepped] = cache[comp ? i : stepped];
					cache[i] = tmp;
				}
				__syncthreads();
			}
		}
		//apply the cache back
		arr[idx] = cache[threadIdx.x];	//this actually is where this value will first be initialised
		g_parents[idx] = parents[threadIdx.x];
		if (idx + nOEMthreads < nnodes)
		{
		arr[idx + nOEMthreads] = cache[threadIdx.x + nOEMthreads];
		g_parents[idx + nOEMthreads] = parents[threadIdx.x + nOEMthreads];
		}
	}

}

__global__ void OddEvenMergeSort_a(int* cache, int* parents, int len, int nnodes)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int section_idx = idx / (len / 2); //there are len/2 comparisons in each block
	int step_idx = idx % (len / 2);
	int i = step_idx + section_idx*len;
	int stepped = i + (len / 2);
	if (stepped < nnodes)
	{
		//compareAndExchangeNodes(cache + i, cache + i + (len / 2));
		bool comp = parents[i] < parents[stepped];
		int tmp = cache[comp ? stepped : i];
		int p = parents[comp ? stepped : i];
		parents[stepped] = parents[comp ? i : stepped];
		parents[i] = p;
		cache[stepped] = cache[comp ? i : stepped];
		cache[i] = tmp;
	}
}

__global__ void OddEvenMergeSort_b(int* cache, int* parents, int len, int nnodes, int step)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int section_idx = idx / (len / 2); //there are len/2 comparisons in each block
	int step_idx = idx % (len / 2);
	int start_idx = step_idx % step;
	int it_idx = step_idx / step;

	int i = section_idx*len + start_idx + step + it_idx * 2 * step;
	int stepped = i + step;
	if ((step * 2 + it_idx * 2 * step < len) && (stepped < nnodes))
	{
		bool comp = parents[i] < parents[stepped];
		int tmp = cache[comp ? stepped : i];
		int p = parents[comp ? stepped : i];
		parents[stepped] = parents[comp ? i : stepped];
		parents[i] = p;
		cache[stepped] = cache[comp ? i : stepped];
		cache[i] = tmp;
	}

}


//temporary helper to check that we have sorted the input
__global__ void displayNodeList(node nodes, int* img, int count)
{
	const int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < count)
	{
		img[id] = nodes.parentind[id];
	}
}


int sortedTo(int * arr, int nvals)
{
	for(int i = 0; i < nvals - 1; i++)
	{
		if (arr[i] < arr[i + 1]) return i;
	}
	return nvals;
}

//this now invalidates .core and .valid in return for better performance
void OEMSort(node arr, int * parents, int nnodes, int rowsxcols)
{
	//allocate the temp array
	 __align__(4) int * ptrs = nullptr;
	try
	{
		g_setToHead <<<nnodes / 1024 + 1, 1024 >>> (arr, parents, rowsxcols, nnodes);

		HANDLE_ERROR(hipMalloc(&ptrs, nnodes * sizeof(int)));
		hipDeviceSynchronize();
		
		HANDLE_ERROR(hipGetLastError());
		OEMSort_kernel <<<nnodes / (nOEMshare) + 1, nOEMthreads >>> (ptrs, parents,  nnodes);
		hipDeviceSynchronize();

		HANDLE_ERROR(hipGetLastError());
		for (int len = nOEMthreads * 2; len <= nnodes; len *= 2)
		{
			OddEvenMergeSort_a <<<nnodes / (nOEMthreads ) + 1, nOEMthreads >>> (ptrs, parents, len, nnodes);
		//	hipDeviceSynchronize();
			for (int step = len / 4; step > 0; step /= 2)
			{
				OddEvenMergeSort_b <<<nnodes / (nOEMthreads) + 1, nOEMthreads >>> (ptrs, parents, len, nnodes, step);
			//	hipDeviceSynchronize();
			}

		}
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());

		applysort <<<nnodes / 64 + 1, 64 >>> (arr, ptrs, parents, nnodes);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());

		if (ptrs != nullptr)HANDLE_ERROR(hipFree(ptrs));
	}
	catch (CUDA_Exception e)
	{
		if(ptrs != nullptr)hipFree(ptrs);
		throw e;
	}
}



//this produces an array of bounding boxes from the sorted nodes
//there are half as many bbs allocated as points
template<int nbbs>
__global__ void makeBBs_local(const node nodes, const float * fg_ranges_min, const float * fg_ranges_max, DidoFusedAnalytics_BoundingBox* bbs, int * bbparents, int * bbnnode,  int rows, int cols)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < rows*cols - 1)
	{
		//zero initialise the values

		__shared__ DidoFusedAnalytics_BoundingBox localbbs[nbbs];
		__shared__ __align__(4) int parents[nbbs];
		__shared__ __align__(4)  int nbbnodes[nbbs];
		if (threadIdx.x % 2 == 0)
		{
			parents[threadIdx.x / 2] = -1;
			nbbnodes[threadIdx.x / 2] = 0;
			localbbs[threadIdx.x / 2].x = 0;
			localbbs[threadIdx.x / 2].y = 0;
			localbbs[threadIdx.x / 2].max_x = 0;
			localbbs[threadIdx.x / 2].max_y = 0;
			localbbs[threadIdx.x / 2].avdepth = 0;
			localbbs[threadIdx.x / 2].uncertainty = 0;
		}

		int myparent = nodes.parentind[index];
		if (myparent >= 0 )
		{
			int myInd = nodes.index[index];
			int myx = myInd % cols;
			int myy = myInd / cols;
			if (myparent == nodes.parentind[index + 1] && ((threadIdx.x == 0) || nodes.parentind[index - 1] != myparent))
			{
				parents[threadIdx.x / 2] = myparent;
				localbbs[threadIdx.x / 2].x = myx;
				localbbs[threadIdx.x / 2].y = myy;
				localbbs[threadIdx.x / 2].max_x = myx;
				localbbs[threadIdx.x / 2].max_y = myy;
			}
			__syncthreads();
			for (int i = 0; i < nbbs; i++)
			{
				if (myparent == parents[i])
				{
					atomicMin(&(localbbs[i].x), myx);
					atomicMin(&(localbbs[i].y), myy);
					atomicMax(&(localbbs[i].max_x), myx);
					atomicMax(&(localbbs[i].max_y), myy);
					if(isfinite(fg_ranges_min[myInd]) && isfinite(fg_ranges_max[myInd])) 
					{
						atomicAdd(&(localbbs[i].avdepth), (fg_ranges_min[myInd] + fg_ranges_max[myInd])/2);
						atomicAdd(&(localbbs[i].uncertainty), (-fg_ranges_min[myInd] + fg_ranges_max[myInd]));
					}
					atomicAdd(&(nbbnodes[i]), 1);
					break;
				}
			}
		}
		__syncthreads();
		//return the values
		if (threadIdx.x % 2 == 0)
		{
			bbs[index / 2] = localbbs[threadIdx.x / 2];
			bbparents[index / 2] = parents[threadIdx.x / 2];
			bbnnode[index / 2] = nbbnodes[threadIdx.x / 2];
		}
	}
}

template __global__ void makeBBs_local<32>(const node nodes, const float * fg_ranges_min, const float * fg_ranges_max, DidoFusedAnalytics_BoundingBox* bbs, int * bbparents, int * bbnnodes, int rows, int cols);


__global__ void downsampleFG(const float * in_pano, float * out_pano, int ncols, int nrows, int scale)
{
	int index_x = (threadIdx.x + blockIdx.x * blockDim.x);
	int index_y = (threadIdx.y + blockIdx.y * blockDim.y);
	__shared__ float totals[16][16];	//the most we could need, for blocks of 32x32 with a scale of 2
	__shared__ int nadded[16][16];
	//work out which total is yours
	int totind_x = threadIdx.x / scale;
	int totind_y = threadIdx.y / scale;
	//initialise the totals
	bool topleft = (threadIdx.x % scale == 0) && (threadIdx.y % scale == 0);
	if (topleft)
	{
		totals[totind_x][totind_y] = 0;
		nadded[totind_x][totind_y] = 0;
	}
	__syncthreads();
	if (index_x / scale < ncols / scale && index_y / scale < nrows / scale)
	{
		if (in_pano[index_x + ncols*index_y] > 0)
		{
			atomicAdd(&totals[totind_x][totind_y], in_pano[index_x + ncols*index_y]);
			//count how many have contributed
			atomicAdd(&nadded[totind_x][totind_y], 1);
		}
		__syncthreads();
		if (topleft)
		{
			//normalise and put the value into the output
			out_pano[(index_x / scale) + (ncols / scale)*(index_y / scale)] =
				nadded[totind_x][totind_y] > 0 ? totals[totind_x][totind_y] / nadded[totind_x][totind_y] : -1.0f;
		}
	}
}



#endif

} //objdetectCUDA

std::vector<DidoFusedAnalytics_BoundingBox> DidoFusedAnalytics_3dObjDetect_CUDA::detectBlobs(const float * foreground_ranges_min, const float * foreground_ranges_max, int rows, int cols) const
{
	std::vector<DidoFusedAnalytics_BoundingBox> rval;
#if DIDOLIDAR_NOGPU

#else

	//auto start = std::chrono::high_resolution_clock::now();
	//allocate local data
	 objdetectCUDA::node m_nodes;
	__align__(16) DidoFusedAnalytics_BoundingBox* m_bbs;
	__align__(4) int * m_parents, *m_nnodes, * tmparray;
	__align__(4) float * d_fgr_min_scaled, * d_fgr_max_scaled;

	//host values
	DidoFusedAnalytics_BoundingBox* l_bbs;
	int * l_parents, *l_nnodes;

	const int blockdim = 32;

	int scaledRows = rows / workingscale;
	int scaledCols = cols / workingscale;

	//a;ways a power of two so we can sort easily
	int nnodes = 1 << int(ceil(log2(scaledRows*scaledCols)));
	int nbbs = (nnodes / 2);

	try
	{
		dim3 grid(((cols) / blockdim + 1), ((rows) / blockdim + 1));
		dim3 block(blockdim, blockdim);
		HANDLE_ERROR(hipMalloc(&d_fgr_min_scaled,scaledRows*scaledCols*sizeof(float) ));
		HANDLE_ERROR(hipMalloc(&d_fgr_max_scaled, scaledRows*scaledCols * sizeof(float)));
		if (workingscale == 1)
		{
			HANDLE_ERROR(hipMemcpy(d_fgr_max_scaled, foreground_ranges_max, scaledRows*scaledCols * sizeof(float), hipMemcpyDeviceToDevice));
			HANDLE_ERROR(hipMemcpy(d_fgr_min_scaled, foreground_ranges_min, scaledRows*scaledCols * sizeof(float), hipMemcpyDeviceToDevice));
		}
		else
		{
			objdetectCUDA::downsampleFG << <grid, block >> > (foreground_ranges_min, d_fgr_min_scaled, cols, rows, workingscale);
			objdetectCUDA::downsampleFG << <grid, block >> > (foreground_ranges_max, d_fgr_max_scaled, cols, rows, workingscale);
		}
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());

		HANDLE_ERROR(hipMalloc(&m_nodes.valid, nnodes * sizeof(bool)));
		HANDLE_ERROR(hipMalloc(&m_nodes.core, nnodes * sizeof(bool)));
		HANDLE_ERROR(hipMalloc(&m_nodes.parentind, nnodes * sizeof(int)));
		HANDLE_ERROR(hipMalloc(&m_nodes.index, nnodes * sizeof(int)));

		grid = dim3(((scaledCols) / blockdim + 1), ((scaledRows) / blockdim + 1));
		block = dim3(blockdim, blockdim);
		objdetectCUDA::generateTrees <<<grid, block >>> (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols);
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());

		
		//	auto gentree = std::chrono::high_resolution_clock::now();

#define SHARED_PDS 0
		HANDLE_ERROR(hipGetLastError());
		switch(epsilon)
		{
		case 1:
#if SHARED_PDS
			objdetectCUDA::PDSDBSCANInit__shared<1, blockdim> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
				hipDeviceSynchronize();

			HANDLE_ERROR(hipGetLastError());
			objdetectCUDA::PDSDBSCANInit__shared<1, blockdim> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
#else
			//less safe version (may be needed?)

			objdetectCUDA::PDSDBSCANInit<1> <<<grid, block >>> (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
#endif
			HANDLE_ERROR(hipMalloc(&tmparray, nnodes * sizeof(int)));
		
			hipDeviceSynchronize();
			HANDLE_ERROR(hipGetLastError());
			//auto pdsinit = std::chrono::high_resolution_clock::now();
			objdetectCUDA::PDSDBSCANLocal<2> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
			break;
		case 2:
#if SHARED_PDS
			objdetectCUDA::PDSDBSCANInit__shared<2, blockdim> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
				hipDeviceSynchronize();

			HANDLE_ERROR(hipGetLastError());
			objdetectCUDA::PDSDBSCANInit__shared<2, blockdim> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
#else
			//less safe version (may be needed?)

			objdetectCUDA::PDSDBSCANInit<2> <<<grid, block >>> (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
#endif
			HANDLE_ERROR(hipMalloc(&tmparray, nnodes * sizeof(int)));
		
			hipDeviceSynchronize();
			HANDLE_ERROR(hipGetLastError());
			//auto pdsinit = std::chrono::high_resolution_clock::now();
			objdetectCUDA::PDSDBSCANLocal<3> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
			break;
		case 3:
#if SHARED_PDS
			objdetectCUDA::PDSDBSCANInit__shared<3, blockdim> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
				hipDeviceSynchronize();

			HANDLE_ERROR(hipGetLastError());
			objdetectCUDA::PDSDBSCANInit__shared<3, blockdim> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
#else
			//less safe version (may be needed?)

			objdetectCUDA::PDSDBSCANInit<3> <<<grid, block >>> (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
#endif
			HANDLE_ERROR(hipMalloc(&tmparray, nnodes * sizeof(int)));
		
			hipDeviceSynchronize();
			HANDLE_ERROR(hipGetLastError());
			//auto pdsinit = std::chrono::high_resolution_clock::now();
			objdetectCUDA::PDSDBSCANLocal<4> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
			break;
		case 4:
#if SHARED_PDS
			objdetectCUDA::PDSDBSCANInit__shared<3, blockdim> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
			hipDeviceSynchronize();

			HANDLE_ERROR(hipGetLastError());
			objdetectCUDA::PDSDBSCANInit__shared<3, blockdim> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
#else
			//less safe version (may be needed?)

			objdetectCUDA::PDSDBSCANInit<4> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
#endif
			HANDLE_ERROR(hipMalloc(&tmparray, nnodes * sizeof(int)));

			hipDeviceSynchronize();
			HANDLE_ERROR(hipGetLastError());
			//auto pdsinit = std::chrono::high_resolution_clock::now();
			objdetectCUDA::PDSDBSCANLocal<5> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
			break;
		case 5:
#if SHARED_PDS
			objdetectCUDA::PDSDBSCANInit__shared<3, blockdim> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
			hipDeviceSynchronize();

			HANDLE_ERROR(hipGetLastError());
			objdetectCUDA::PDSDBSCANInit__shared<3, blockdim> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
#else
			//less safe version (may be needed?)

			objdetectCUDA::PDSDBSCANInit<5> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
#endif
			HANDLE_ERROR(hipMalloc(&tmparray, nnodes * sizeof(int)));

			hipDeviceSynchronize();
			HANDLE_ERROR(hipGetLastError());
			//auto pdsinit = std::chrono::high_resolution_clock::now();
			objdetectCUDA::PDSDBSCANLocal<5> << <grid, block >> > (d_fgr_min_scaled, d_fgr_max_scaled, m_nodes, scaledRows, scaledCols, rangeScaling, ncore);
			break;
		default:
			throw "only integer eps of 1 to 5 supported";
		}
		HANDLE_ERROR(hipMalloc(&m_bbs, nbbs * sizeof(DidoFusedAnalytics_BoundingBox)));
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());

	//	auto pdslocal = std::chrono::high_resolution_clock::now();
		//sort the nodes by parent
		objdetectCUDA::OEMSort(m_nodes, tmparray, nnodes, scaledRows*scaledCols);
		HANDLE_ERROR(hipMalloc(&m_parents, nbbs * sizeof(int)));
		HANDLE_ERROR(hipMalloc(&m_nnodes, nbbs * sizeof(int)));
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());
	//	auto oesort = std::chrono::high_resolution_clock::now();
		
		//make blobs from them

		objdetectCUDA::makeBBs_local<32> <<< nnodes / 64 + 1, 64 >>> (m_nodes, d_fgr_min_scaled, d_fgr_max_scaled, m_bbs, m_parents, m_nnodes, scaledRows, scaledCols);

		l_parents = (int*)malloc(nbbs * sizeof(int));
		l_nnodes = (int*)malloc(nbbs * sizeof(int));
		l_bbs = (DidoFusedAnalytics_BoundingBox*)malloc(nbbs * sizeof(DidoFusedAnalytics_BoundingBox));
		hipDeviceSynchronize();
		HANDLE_ERROR(hipGetLastError());
	//	auto makeb = std::chrono::high_resolution_clock::now();

		//copy down the data
		HANDLE_ERROR(hipMemcpy(l_parents, m_parents, nbbs * sizeof(int), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(l_nnodes, m_nnodes, nbbs * sizeof(int), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(l_bbs, m_bbs, nbbs * sizeof(DidoFusedAnalytics_BoundingBox), hipMemcpyDeviceToHost));

		int prevparent = -1;
		int prevnnodes = 0;
		for (int i = 0; i < nbbs ; i++)
		{
			if (l_nnodes[i] == 0) continue; //skip empty entries
			//breaks once we hit the -1s
			if (l_parents[i] < 0)
			{
				break;
			}

			//check if it was split
			if (l_parents[i] == prevparent)
			{
				rval.back().x = std::min(rval.back().x, l_bbs[i].x);
				rval.back().y = std::min(rval.back().y, l_bbs[i].y);
				rval.back().max_x = std::max(rval.back().max_x, l_bbs[i].max_x);
				rval.back().max_y = std::max(rval.back().max_y, l_bbs[i].max_y);
				rval.back().avdepth += l_bbs[i].avdepth;
				rval.back().uncertainty += l_bbs[i].uncertainty;
				prevnnodes += l_nnodes[i];
			}
			///else append to output
			else
			{
				if (!rval.empty())
				{
					//only keep ones that are big enough
					if (prevnnodes > minpoints)
					{
						rval.back().avdepth /= prevnnodes;
						rval.back().uncertainty /= prevnnodes;
						rval.back().x *= workingscale;
						rval.back().max_x *= workingscale;
						rval.back().y *= workingscale;
						rval.back().max_y *= workingscale;
						if(rval.back().x < 0 || rval.back().x > cols || rval.back().y < 0 || rval.back().y > rows ||
						rval.back().max_x < 0 || rval.back().max_x > cols || rval.back().max_y < 0 || rval.back().max_y > rows)
							rval.pop_back(); //sanity check
					}
					else rval.pop_back();

				}
				rval.push_back(l_bbs[i]);
				prevparent = l_parents[i];
				prevnnodes = l_nnodes[i];
			}
		}
		if (!rval.empty())
		{
			if (prevnnodes > minpoints)
			{
				rval.back().avdepth /= prevnnodes;
				rval.back().uncertainty /= prevnnodes;
				rval.back().x *= workingscale;
				rval.back().max_x *= workingscale;
				rval.back().y *= workingscale;
				rval.back().max_y *= workingscale;

				if (rval.back().x < 0 || rval.back().x > cols || rval.back().y < 0 || rval.back().y > rows ||
					rval.back().max_x < 0 || rval.back().max_x > cols || rval.back().max_y < 0 || rval.back().max_y > rows)
					rval.pop_back(); //sanity check
			}
			else
			{
				rval.pop_back();
			}
		}

	//	auto colb = std::chrono::high_resolution_clock::now();


		if (l_nnodes != nullptr) free(l_nnodes);
		if (l_bbs != nullptr) free(l_bbs);
		if (l_parents != nullptr) free(l_parents);

		//free local data
		if (d_fgr_min_scaled != nullptr) HANDLE_ERROR(hipFree(d_fgr_min_scaled));
		if (d_fgr_max_scaled != nullptr) HANDLE_ERROR(hipFree(d_fgr_max_scaled));
		if (tmparray != nullptr) HANDLE_ERROR(hipFree(tmparray));
		if (m_nodes.valid != nullptr) HANDLE_ERROR(hipFree(m_nodes.valid));
		if (m_nodes.core != nullptr) HANDLE_ERROR(hipFree(m_nodes.core));
		if (m_nodes.parentind != nullptr) HANDLE_ERROR(hipFree(m_nodes.parentind));
		if (m_nodes.index != nullptr) HANDLE_ERROR(hipFree(m_nodes.index));
		if (m_nnodes != nullptr) HANDLE_ERROR(hipFree(m_nnodes));
		if (m_bbs != nullptr) HANDLE_ERROR(hipFree(m_bbs));
		if (m_parents != nullptr) HANDLE_ERROR(hipFree(m_parents));

	}
	catch (CUDA_Exception e)
	{
		if (l_nnodes != nullptr) free(l_nnodes);
		if (l_bbs != nullptr) free(l_bbs);
		if (l_parents != nullptr) free(l_parents);
		//free local data in case of error and reset the context
		hipDeviceReset();
		throw e;
	}
#endif

	//populate it

	return rval;
}
}